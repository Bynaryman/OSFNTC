#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/NumericUtils.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorCompare.h>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/core/Scalar.h>


namespace at { namespace native {

namespace {

void where_kernel_impl(TensorIterator &iter, ScalarType condition_type) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBFloat16, kBool, iter.dtype(), "where_cuda", [&] {
    if (condition_type == at::ScalarType::Byte) {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (uint8_t cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    } else {
      gpu_kernel(
        iter,
        [=] GPU_LAMBDA (bool cond_val, scalar_t self_val, scalar_t other_val) -> scalar_t {
          return cond_val ? self_val : other_val;
        });
    }
  });
}

void isposinf_kernel_impl(TensorIteratorBase &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isposinf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void isneginf_kernel_impl(TensorIteratorBase &iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.input_dtype(), "isneginf_cuda", [&]() {
    gpu_kernel(
      iter,
      [] GPU_LAMBDA (scalar_t a) -> bool { return a == -std::numeric_limits<scalar_t>::infinity(); }
    );
  });
}

void clamp_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_cuda", [&] {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t v, scalar_t lower, scalar_t upper) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (at::_isnan(v)) {
        return v;
      } else {
        return ::min(::max(v, lower), upper);
      }
    });
  });
}

void clamp_scalar_kernel_impl(TensorIteratorBase& iter, const Scalar& min, const Scalar& max) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_scalar_cuda", [&] {
    const auto lower = min.to<scalar_t>();
    const auto upper = max.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (at::_isnan(v)) {
        return v;
      } else {
        return ::min(::max(v, lower), upper);
      }
    });
  });
}

void clamp_min_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_min_cuda", [&] {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t v, scalar_t lower) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::max(v, lower);
      }
    });
  });
}

void clamp_min_scalar_kernel_impl(TensorIterator& iter, Scalar min) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_min_scalar_cuda", [&] {
    auto lower = min.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::max(v, lower);
      }
    });
  });
}

void clamp_max_kernel_impl(TensorIterator& iter) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_max_cuda", [&] {
    gpu_kernel_with_scalars(iter, []GPU_LAMBDA(scalar_t v, scalar_t upper) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(v, upper);
      }
    });
  });
}

void clamp_max_scalar_kernel_impl(TensorIterator& iter, Scalar max) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "clamp_max_scalar_cuda", [&] {
    const auto upper = max.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(v, upper);
      }
    });
  });
}

} // anonymous namespace


REGISTER_DISPATCH(where_kernel, &where_kernel_impl);
REGISTER_DISPATCH(isposinf_stub, &isposinf_kernel_impl);
REGISTER_DISPATCH(isneginf_stub, &isneginf_kernel_impl);
REGISTER_DISPATCH(clamp_stub, &clamp_kernel_impl);
REGISTER_DISPATCH(clamp_min_stub, &clamp_min_kernel_impl);
REGISTER_DISPATCH(clamp_max_stub, &clamp_max_kernel_impl);
REGISTER_DISPATCH(clamp_scalar_stub, &clamp_scalar_kernel_impl);
REGISTER_DISPATCH(clamp_min_scalar_stub, &clamp_min_scalar_kernel_impl);
REGISTER_DISPATCH(clamp_max_scalar_stub, &clamp_max_scalar_kernel_impl);

template <typename scalar_t>
__global__ void _assert_async_cuda_kernel(scalar_t* input) {
  CUDA_KERNEL_ASSERT(input[0] != 0);
}

__global__ void _assert_async_cuda_kernel(c10::complex<float>* input) {
  CUDA_KERNEL_ASSERT(input[0] != c10::complex<float>(0, 0));
}
__global__ void _assert_async_cuda_kernel(c10::complex<double>* input) {
  CUDA_KERNEL_ASSERT(input[0] != c10::complex<double>(0, 0));
}

void _assert_async_cuda(const Tensor& self_tensor) {
  const TensorBase &self = get_tensor_base(self_tensor);
  auto n = self.numel();
  TORCH_CHECK(n != 0, "Boolean value of Tensor with no values is ambiguous");
  TORCH_CHECK(n < 2, "Boolean value of Tensor with more than one value is ambiguous");
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16, self.scalar_type(), "_assert_async_cuda", [&] {
    _assert_async_cuda_kernel<<<1, 1, 0, stream>>>(self.data_ptr<scalar_t>());
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

}} // namespace at::native
