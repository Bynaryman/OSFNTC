#include "hip/hip_runtime.h"
#include <ATen/native/TensorAdvancedIndexing.h>
#include <ATen/native/IndexingUtils.h>

#include <ATen/ATen.h>
#include <ATen/ceil_div.h>
#include <ATen/NativeFunctions.h>
#include <ATen/ExpandUtils.h>
#include <ATen/MemoryOverlap.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/Resize.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/CUDAUtils.h>

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/cub.h>
#include <c10/util/irange.h>
#include <c10/core/QScheme.h>

#include <limits>

#include <c10/macros/Macros.h>

namespace {

template <typename scalar_t, int SZ>
__global__ void indexing_backward_kernel(
  int64_t* sorted_indices, int64_t* indices, scalar_t* grad_output, scalar_t* grad_weight,
  int64_t numel, int64_t stride, int64_t stride_before, int64_t outer_dim, bool accumulate) {
//numel is total number of flattened indices, not expanded to dimensions that are not indexed.
//stride is the cumulative size of the not-indexed last dimensions
//stride_before is the stride of the dimension immediately preceding first indexed dimension
//if indexing starts from the 0th dimension, stride_before does not matter because blockIdx.z will be 0 in this case
//outer_dim is number of elements in the first unindexed dimensions
  using accscalar_t = at::acc_type<scalar_t, true>;

  // Each warp is responsible for an input into the LookupTable.
  // If the preceding input has the same destination index as this input, then the warp
  // exits immediately. The warp also processes subsequent inputs with the
  // same value.
  //
  // Input Warp
  // 1     <warp 1>
  // 1     <warp 1> (<warp 2> exits without doing any work)
  // 5     <warp 3>
  // 8     <warp 4>

  // Number of values processed by each thread (grain size)
  for (int64_t z = blockIdx.z; z < outer_dim; z += gridDim.z){
    int64_t idx = blockIdx.x * blockDim.y + threadIdx.y;
    if (idx < numel
        && (idx == 0 || sorted_indices[idx] != sorted_indices[idx - 1])){
      do {
        int64_t start_feature = threadIdx.x + blockIdx.y * blockDim.x * SZ;
        // if not accumulate, we only keep the last duplicate index so skip those before it
        if (!accumulate && (idx < numel - 1) && sorted_indices[idx] == sorted_indices[idx + 1]) {
          idx++;
          continue;
        }
        const int64_t weight_row = ((int64_t) sorted_indices[idx]) * stride + z * stride_before;
        const int64_t grad_row = ((int64_t) indices[idx]) * stride + z * numel * stride;
        const accscalar_t scale = (accscalar_t)1.0;

        accscalar_t gradient[SZ];
        accscalar_t weight[SZ];

        while (start_feature < stride) {
          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
              gradient[ii] = static_cast<accscalar_t>(grad_output[grad_row + feature_dim]);
              if (accumulate) {
                weight[ii] = static_cast<accscalar_t>(grad_weight[weight_row + feature_dim]);
              }
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            if (accumulate) {
              weight[ii] += gradient[ii] * scale;
            } else {
              weight[ii] = gradient[ii] * scale;
            }
          }

          #pragma unroll
          for (int ii = 0; ii < SZ; ii++) {
            int64_t feature_dim = start_feature + ii * C10_WARP_SIZE;
            if (feature_dim < stride) {
                grad_weight[weight_row + feature_dim] = static_cast<scalar_t>(weight[ii]);
            }
          }
          start_feature += gridDim.y * blockDim.x * SZ;
        }

        idx++;
      } while (idx < numel && sorted_indices[idx] == sorted_indices[idx - 1]);
    }
  }
}


}


namespace at { namespace native {

static Tensor wrapIndexOnce(const Tensor & index, int64_t dim, int64_t dim_size, bool check_range=true) {
//we don't need to check range in backward - if there were out of bounds indices forward should already have errored out
  if (index.numel() != 0 && check_range) {
    auto max_idx = index.max().item<int64_t>();
    auto min_idx = index.min().item<int64_t>();
    if (max_idx >= dim_size) {
      TORCH_CHECK_INDEX(false, "index ", max_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
    if (min_idx < -dim_size) {
      TORCH_CHECK_INDEX(false, "index ", min_idx, " is out of bounds for dimension ", dim, " with size ", dim_size);
    }
  }
  return index.remainder(dim_size);
}

static std::vector<int64_t> computeLinearStride(const Tensor & tensor) {
  // computes the stride as if tensor were contiguous
  auto sizes = tensor.sizes();
  std::vector<int64_t> stride(tensor.dim());
  stride[tensor.dim() - 1] = 1;
  std::partial_sum(sizes.rbegin(), sizes.rend() - 1, stride.rbegin() + 1, std::multiplies<int64_t>());
  return stride;
}

static std::tuple<Tensor, int64_t, int64_t, int64_t>
computeLinearIndex(const Tensor & src, TensorList indices, bool check_range) {
  auto strides = computeLinearStride(src);
  const auto& device = src.options().device();

  // Compute the linear index by multiplying the indexing tensors by the
  // stride and summing them. All the indexing tensors have the same shape at
  // this point. We also compute the number of dimensions before and after that
  // are not being index.
  Tensor linearIndex;
  int64_t emptyBefore = 0, emptyAfter = 0, nElemBefore = 1, nElemAfter = 1, strideBefore =0;
  for (const auto i: c10::irange(src.dim())) {
    if (indices[i].defined()) {
      // Cast index to the longType matching src's device
      // This allows us to support ie indexing a cuda tensor with a cpu tensor
      Tensor index = (wrapIndexOnce(indices[i], i, src.size(i), check_range) * strides[i]).to(device);
      if (linearIndex.defined()) {
        linearIndex += index;
      } else {
        linearIndex = index;
        if (i>0) {
           strideBefore = src.stride(i-1); // stride after undefined dimensions
        }
      }
    } else if (linearIndex.defined()) {
      emptyAfter++;
      nElemAfter *= src.size(i);
    } else {
      emptyBefore++;
      nElemBefore *= src.size(i);
    }
  }

  return std::make_tuple(std::move(linearIndex), nElemBefore, strideBefore, nElemAfter);
}


static std::tuple<Tensor, Tensor, int64_t, int64_t, int64_t, std::vector<int64_t>> makeLinearIndex(Tensor self, const c10::List<c10::optional<at::Tensor>>& orig, bool check_range) {
  checkIndexTensorTypes(orig);
  // first expand BoolTensor (masks) or ByteTensor (masks) into 1 or more LongTensors
  auto indices = expandTensors(self, orig);
  // next broadcast all index tensors together
  indices = expand_outplace(indices);
  // add missing null Tensors so that it matches self.dim()
  while (indices.size() < (size_t)self.dim()) {
    indices.emplace_back();
  }
  // if the non-null indices are not all adjacent, transpose self and indices
  // together so that they're adjacent at the front
  std::vector<int64_t> inversePerm;
  if (!hasContiguousSubspace(indices)) {
    std::tie(self, indices, inversePerm) = transposeToFrontAndInvPerm(self, indices);
  }
  int64_t nElemBefore, strideBefore, nElemAfter;
  Tensor linearIndex;
  std::tie(linearIndex, nElemBefore, strideBefore, nElemAfter) = computeLinearIndex(self, indices, check_range);
  return std::make_tuple(linearIndex, self, nElemBefore, strideBefore, nElemAfter, inversePerm);
}


void index_put_with_sort_kernel_thrust_helper(Tensor &linearIndex, Tensor &orig_indices, Tensor &sorted_indices, int64_t num_indices);

namespace {

int64_t largestIndex(const Tensor &self) {
  int64_t result = 0;
  for (const auto i: c10::irange(self.dim())) {
    result += (self.sizes()[i] - 1) * self.strides()[i];
  }
  return result;
}

void index_put_with_sort_kernel(Tensor & self, const c10::List<c10::optional<Tensor>>& indices, const Tensor & value, bool accumulate, bool unsafe) {
  if (indices.size() > (size_t)self.dim()) {
    TORCH_CHECK_INDEX(false, "too many indices for tensor of dimension ", self.dim(), " (got ", indices.size(), ")");
  }
  if (!self.is_contiguous()) {
    self = self.contiguous();
  }
  Tensor linearIndex, src, expandedValue = value;
  int64_t nElemBefore, strideBefore, sliceSize;
  std::vector<int64_t> inversePerm;
  std::tie(linearIndex, src, nElemBefore, strideBefore, sliceSize, inversePerm) = makeLinearIndex(self, indices, !unsafe);
  int64_t num_indices = linearIndex.numel();

  if (expandedValue.numel() < num_indices * nElemBefore * sliceSize) {
    auto expanded_size = at::DimVector(expandedValue.sizes());
    auto size1 = expandedValue.sizes();
    auto size2 = linearIndex.sizes();
    if (are_expandable(size1, size2)) {
      expanded_size = infer_size_dimvector(size1, size2);
    }
    if (nElemBefore > 1) {
      expanded_size.insert(expanded_size.begin(), nElemBefore);
    }
    expandedValue = expandedValue.expand(expanded_size);
  }
  expandedValue = expandedValue.contiguous();

  if (num_indices > 0 && sliceSize > 0) {
      const bool permuted = !src.is_contiguous();
      auto src_ = permuted ? src.contiguous() : src;
      linearIndex = linearIndex.reshape(-1);
      auto sorted_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      auto orig_indices = at::empty_like(linearIndex, LEGACY_CONTIGUOUS_MEMORY_FORMAT);
      const hipStream_t stream = at::cuda::getCurrentCUDAStream();

      linearIndex.divide_(sliceSize, "trunc");

      // cub on CUDA <= 11.2 have a bug that for small sizes
      // cub's sort can be much slower than thrust's merge sort
      // this bug is fixed in CUDA 11.3
#if (defined(CUDA_VERSION) && CUDA_VERSION < 11030) || defined(USE_ROCM)
      if (num_indices < 50000) {
        index_put_with_sort_kernel_thrust_helper(linearIndex, orig_indices, sorted_indices, num_indices);
      } else
#endif
      {
      // Sort the inputs into sorted with the corresponding indices
      auto range = at::arange(num_indices, linearIndex.options());
      // linearIndex can not be negative, and we take advantage of this
      // fact to sort on less bits for better performance.
      int64_t nbits = cuda::hipcub::get_num_bits(largestIndex(self) / sliceSize);
      cuda::hipcub::radix_sort_pairs(
        linearIndex.data_ptr<int64_t>(), sorted_indices.data_ptr<int64_t>(),
        range.data_ptr<int64_t>(), orig_indices.data_ptr<int64_t>(),
        num_indices, false, 0, nbits);
      }

      TORCH_INTERNAL_ASSERT(
          linearIndex.numel()*sliceSize*nElemBefore == expandedValue.numel(),
          "number of flattened indices did not match number of elements in the value tensor: ",
          linearIndex.numel()*sliceSize*nElemBefore, " vs ", expandedValue.numel());
      const int UNROLL = 4;
      const int indices_per_block = 4;
      dim3 grid(ceil_div(num_indices, (int64_t) indices_per_block),
           std::min<int>(at::cuda::getCurrentDeviceProperties()->maxGridSize[1], ceil_div(sliceSize, (int64_t) (C10_WARP_SIZE*UNROLL))),
           std::min(std::max<int>(1,nElemBefore), at::cuda::getCurrentDeviceProperties()->maxGridSize[2]));
      dim3 block(C10_WARP_SIZE, indices_per_block);

      AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Half, at::ScalarType::Bool, at::ScalarType::BFloat16,
      expandedValue.scalar_type(), "indexing_backward", [&] {
        indexing_backward_kernel<scalar_t, UNROLL><<<grid, block, 0, stream>>>(
          sorted_indices.data_ptr<int64_t>(),
          orig_indices.data_ptr<int64_t>(),
          expandedValue.data_ptr<scalar_t>(),
          src_.data_ptr<scalar_t>(),
          num_indices,
          sliceSize,
          strideBefore,
          nElemBefore,
          accumulate);
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });

      if (permuted) {
        self.copy_(src_.permute(inversePerm));
      }
  }
}

REGISTER_CUDA_DISPATCH(index_put_with_sort_stub, &index_put_with_sort_kernel);
} //anonymous


// Check tensor dimensions for index operations, and return the slice size.
static ptrdiff_t getSliceSize(const Tensor & dst,
                              int dim,
                              const Tensor & index,
                              const Tensor & src)
{
  const auto dstDims = dst.dim();
  const auto srcDims = src.dim();

  TORCH_CHECK(index.dim() <= 1, "Index must be vector or scalar");

  ptrdiff_t dstSliceSize = 1;
  TORCH_CHECK(dim >= 0 && dim < dstDims, "Indexing dim ", dim, " is out of bounds");
  for (const auto d: c10::irange(dstDims)) {
    if (d != dim) {
      dstSliceSize *= dst.size(d);
    }
  }

  TORCH_CHECK(dim < srcDims, "Indexing dim ", dim, " is out of bounds");
  TORCH_CHECK(index.numel() == src.size(dim),
             "length of src.size[dim] is not equal to length of indices");

  ptrdiff_t srcSliceSize = 1;
  bool mismatch = false;

  if (dstDims != srcDims) mismatch = true;

  for (const auto d: c10::irange(srcDims)) {
    if (d != dim) {
      srcSliceSize *= src.size(d);
      if (!mismatch && dst.size(d) != src.size(d)) mismatch = true;
    }
  }

  TORCH_CHECK(dstSliceSize == srcSliceSize,
             "Source/destination tensor have different slice sizes (%ld vs %ld)",
             dstSliceSize, srcSliceSize);

  if (mismatch) {
    TORCH_WARN_ONCE(
        "Warning: source/destination slices have same size but different "
        "shape for an index operation.  This behavior is deprecated.\n");
  }

  return dstSliceSize;
}

// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexAddLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexAddSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize,
                                   T alpha) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType srcIndex = 0; srcIndex < indices.sizes[0]; ++srcIndex) {
    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
          cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstAddDim];

      IndexType srcOffset =
          cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcAddDim];

      gpuAtomicAddNoReturn(&dst.data[dstOffset], src.data[srcOffset] * alpha);
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexAddSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexAddLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                   cuda::detail::TensorInfo<T, IndexType> src,
                                   cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                   int dstAddDim,
                                   int srcAddDim,
                                   IndexType totalSize,
                                   IndexType innerSize,
                                   int64_t dstAddDimSize,
                                   T alpha) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType srcIndex, elementInSlice;
    if (IndexIsMajor) {
      srcIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      srcIndex = linearIndex % innerSize;
    }

    // Lua indices begin at 1
    IndexType dstIndex =
        indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(srcIndex, indices)];
    CUDA_KERNEL_ASSERT(dstIndex < dstAddDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstAddDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcAddDim];

    gpuAtomicAddNoReturn(&dst.data[dstOffset], src.data[srcOffset] * alpha);
  }
}

// Compare the stride between adjacent slices (sliceStride) with strides in the
// other dimensions (i.e., strides *inside* each slice).
//
// - Returns true if some dimension inside the slice has lower stride than
//   sliceStride.  The simplest example is a 2-D contiguous tensor with sliceDim
//   == 0 (that is, each slice is a row).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "index-major order".  For example, if thread count equals slice size, then
//   all threads process slice #0 in lockstep, and then slice #1, and so on.
//
// - Otherwise (i.e., sliceStride has the lowest value), this function returns
//   false.  The simplest example is a 2-D contiguous tensor with sliceDim == 1
//   (each slice is a column).
//
//   In this case, we choose the CUDA kernel that processes the data in
//   "elementInSlice-major order".  For example, each thread can process element
//   #0 of every slice, and then element #1 of every slice, and so on.
template <typename scalar_t>
bool indexShouldBeMajor(cuda::detail::TensorInfo<scalar_t, unsigned int> &info,
                                    int sliceDim)
{
  // The stride between adjacent slices (e.g., between element #0 of slice #100
  // and element #0 of slice #101).
  unsigned int sliceStride = info.strides[sliceDim];

  for (const auto i: c10::irange(info.dims)) {
    if (i != sliceDim && info.sizes[i] > 1 && info.strides[i] < sliceStride) {
      return true;
    }
  }

  return false;
}

Tensor& index_add_cuda_(Tensor & self, int64_t dim, const Tensor & index, const Tensor & source, const Scalar &alpha) {
  dim = maybe_wrap_dim(dim, self.dim());

  TensorArg self_arg{self, "self", 1}, index_arg{index, "index", 3}, source_arg{source, "source", 4};
  checkAllSameGPU(__func__, {self_arg, index_arg, source_arg});

  TORCH_CHECK_INDEX(index.dim() <= 1, "index_add_(): Index is supposed to be a vector");
  TORCH_CHECK(index.scalar_type() == ScalarType::Long || index.scalar_type() == ScalarType::Int, "index_add_(): Expected dtype int32/int64 for index");
  TORCH_CHECK(self.scalar_type() == source.scalar_type(),
              "index_add_(): self and source must have the same scalar type");
  TORCH_CHECK(dim == 0 || dim < source.dim(),
              "index_add_(): Indexing dim ", dim, " is out of bounds of tensor");
  TORCH_CHECK(index.numel() == (source.dim() == 0 ? 1 : source.size(dim)),
              "index_add_(): Number of indices should be equal to self.size(dim)");

  at::assert_no_internal_overlap(self);
  at::assert_no_overlap(self, index);
  at::assert_no_overlap(self, source);

  // Scalars are treated as 1-d tensor
  Tensor self_ = (self.dim() == 0) ? self.view(1) : self;
  Tensor source_ = (source.dim() == 0) ? source.view(1) : source;

  TORCH_CHECK(self.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");
  TORCH_CHECK(source.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims" );
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, "tensor has too many (>", MAX_TENSORINFO_DIMS, ") dims");

  at::assert_no_internal_overlap(self);
  at::assert_no_partial_overlap(self, index);
  at::assert_no_partial_overlap(self, source);

  if (globalContext().deterministicAlgorithms()){
    torch::List<c10::optional<Tensor>> indices;
    indices.reserve(dim + 1);
    for (const auto i: c10::irange(dim)) {
      indices.emplace_back();
    }
    indices.emplace_back(index.to(at::kLong));
    return self.index_put_(indices, source * alpha, true);
  }

  // The `source` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of index we are choosing, which is the total size
  // of the tensor `index`.
  ptrdiff_t sliceSize = getSliceSize(self_, dim, index, source_);
  ptrdiff_t sourceTotalSize = source.numel();
  int64_t selfAddDimSize = self_.size(dim);
  ptrdiff_t numIndex = index.numel();

  if (sliceSize == 0) {
    return self;
  }
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  bool indContig = index.is_contiguous();

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM)  \
  indexAddSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, SELF_DIM, SOURCE_DIM, IDX_DIM> \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                \
      selfInfo, sourceInfo, indexInfo,                                               \
      selfAddDim, sourceAddDim, sliceSize, selfAddDimSize, alpha_value);             \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                        \
                    SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR)            \
  indexAddLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                       \
                     SELF_DIM, SOURCE_DIM, IDX_DIM, IDX_IS_MAJOR>           \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                       \
      selfInfo, sourceInfo, indexInfo,                                      \
      selfAddDim, sourceAddDim, sourceTotalSize,                            \
      (IDX_IS_MAJOR) ? sliceSize : numIndex,                                \
      selfAddDimSize, alpha_value);                                         \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(ceil_div(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(ceil_div(sourceTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(sourceTotalSize, (ptrdiff_t)128));

  if (cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(source) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, unsigned int> selfInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(self_);
      int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);
      auto alpha_value = alpha.to<scalar_t>();
      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        auto sourceInfo =
          cuda::detail::getTensorInfo<scalar_t, unsigned int>(source_);
        int sourceAddDim = sourceInfo.collapseDims(dim);
        sourceInfo.reduceDim(sourceAddDim);

        auto indexInfo =
        cuda::detail::getTensorInfo<index_t, unsigned int>(index);
        indexInfo.collapseDims();

        // A reasonable choice for when to have each thread iterate over
        // index to choose
        if (numIndex <= 16) {
          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
          } else {
            SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
          }
        } else {
          bool indexIsMajor = indexShouldBeMajor(selfInfo, selfAddDim);

          if (selfInfo.dims == 1 && sourceInfo.dims == 1 && indContig) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
          } else if (selfInfo.dims == 2 && sourceInfo.dims == 2 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
            }
          } else if (selfInfo.dims == 3 && sourceInfo.dims == 3 && indContig) {
            if (indexIsMajor) {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
            } else {
              LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
            }
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
          }
        }
      });
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(at::ScalarType::Bool, at::ScalarType::Half, at::ScalarType::BFloat16, self.scalar_type(), "index_add", [&] {
      cuda::detail::TensorInfo<scalar_t, uint64_t> selfInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(self_);
      int selfAddDim = selfInfo.collapseDims(dim);
      selfInfo.reduceDim(selfAddDim);
      auto alpha_value = alpha.to<scalar_t>();

      cuda::detail::TensorInfo<scalar_t, uint64_t> sourceInfo =
        cuda::detail::getTensorInfo<scalar_t, uint64_t>(source_);
      int sourceAddDim = sourceInfo.collapseDims(dim);
      sourceInfo.reduceDim(sourceAddDim);

      AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_add_cuda_", [&] () {
        cuda::detail::TensorInfo<index_t, uint64_t> indexInfo =
          cuda::detail::getTensorInfo<index_t, uint64_t>(index);
        indexInfo.collapseDims();

        LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
      });
    });
  }

  return self;
#undef SMALL_INDEX
#undef LARGE_INDEX
}

namespace {
// We prefer this kernel to avoid reloading index points if the number
// of indices is a small number.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is large, then the
// indexSelectLargeIndex kernel is a better choice to increase
// parallelism.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim>
__global__ void indexSelectSmallIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<T, IndexType> src,
                                      cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // In order to avoid reloading the index that we are copying, load
  // it once to handle all of the points that are being selected, so
  // it can be reused as much as possible. This kernel is chosen when
  // this is a good choice (small number of chosen indices), since
  // re-accessing indices in addition to src elements can be slow.
  for (IndexType dstIndex = 0; dstIndex < indices.sizes[0]; ++dstIndex) {
    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    // We stride over the output ignoring the indexed dimension
    // (innerSize), whose offset calculation is handled differently
    for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
         linearIndex < innerSize;
         linearIndex += gridDim.x * blockDim.x) {
      IndexType dstOffset =
        cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(linearIndex, dst);
      dstOffset += dstIndex * dst.strides[dstSelectDim];

      IndexType srcOffset =
        cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(linearIndex, src);
      srcOffset += srcIndex * src.strides[srcSelectDim];

      dst.data[dstOffset] = src.data[srcOffset];
    }
  }
}

// We prefer this kernel to balance parallelism across index points,
// if there are a large number of indices.
// This kernel in fact works for all choices of problem size, but if
// the number of indices chosen is small, then the
// indexSelectSmallIndex kernel is a better choice to reduce memory
// accesses.
template <typename T, typename IndicesType, typename IndexType, int DstDim, int SrcDim, int IdxDim,
          bool IndexIsMajor>
__global__ void indexSelectLargeIndex(cuda::detail::TensorInfo<T, IndexType> dst,
                                      cuda::detail::TensorInfo<T, IndexType> src,
                                      cuda::detail::TensorInfo<IndicesType, IndexType> indices,
                                      int dstSelectDim,
                                      int srcSelectDim,
                                      IndexType totalSize,
                                      IndexType innerSize,
                                      int64_t srcSelectDimSize) {
  // We stride over the output including the indexed dimension
  // (totalSize), and calculate the destination index point based on that
  for (IndexType linearIndex = blockIdx.x * blockDim.x + threadIdx.x;
       linearIndex < totalSize;
       linearIndex += gridDim.x * blockDim.x) {
    IndexType dstIndex, elementInSlice;
    if (IndexIsMajor) {
      dstIndex = linearIndex / innerSize;
      elementInSlice = linearIndex % innerSize;
    }
    else {
      elementInSlice = linearIndex / innerSize;
      dstIndex = linearIndex % innerSize;
    }

    IndexType srcIndex =
      indices.data[cuda::detail::IndexToOffset<IndicesType, IndexType, IdxDim>::get(dstIndex, indices)];
    CUDA_KERNEL_ASSERT(srcIndex < srcSelectDimSize);

    IndexType dstOffset =
      cuda::detail::IndexToOffset<T, IndexType, DstDim>::get(elementInSlice, dst);
    dstOffset += dstIndex * dst.strides[dstSelectDim];

    IndexType srcOffset =
      cuda::detail::IndexToOffset<T, IndexType, SrcDim>::get(elementInSlice, src);
    srcOffset += srcIndex * src.strides[srcSelectDim];

    dst.data[dstOffset] = src.data[srcOffset];
  }
}

namespace {

// When using a 0-dim scalar tensor, we need the legacy (THC) semantics of
// TensorInfo: Pretend that the scalar tensor is in fact a one-element vector.
template <typename T, typename IndexType>
cuda::detail::TensorInfo<T, IndexType>
tensorInfoLegacyIfScalar(cuda::detail::TensorInfo<T, IndexType> ti) {
  if (ti.dims == 0) {
    ti.dims = 1;
    ti.sizes[0] = 1;
    ti.strides[0] = 1;
  }
  return ti;
}

}

template <typename scalar_t>
void index_select_out_cuda_impl(
    Tensor& out,
    const Tensor& self,
    long dim,
    const Tensor& index) {
  ptrdiff_t numIndices = index.numel();
  int selfDims = self.dim() == 0 ? 1 : self.dim();

  const hipStream_t stream = at::cuda::getCurrentCUDAStream();

  TORCH_CHECK(
      index.dim() <= 1, "Index is supposed to be an empty tensor or a vector");
  TORCH_CHECK(dim < selfDims, "Indexing dim is out of bounds");

  std::vector<int64_t> newSize = self.sizes().vec();
  if (self.dim() > 0) {
    newSize[dim] = numIndices;
  }

  if (self.is_quantized()){
      out = at::empty_quantized(newSize, out);
  } else {
    at::native::resize_output(out, newSize);
  }

  ptrdiff_t outTotalSize = out.numel();
  if (outTotalSize == 0) {
    return;
  }

  bool indContig = index.is_contiguous();

  // The `self` is partitioned into two parts:
  // -the size of each slice we are indexing, which is the
  // total size of the tensor ignoring dimension `dim`;
  // -the number of indices we are choosing, which is the total size
  // of the tensor `indices`.
  int64_t selfSelectDimSize = self.dim() == 0 ? 1 : self.size(dim);
  ptrdiff_t sliceSize = outTotalSize / numIndices;

  int mpc = at::cuda::getCurrentDeviceProperties()->multiProcessorCount;

#define SMALL_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM)         \
  indexSelectSmallIndex<TENSOR_TYPE, INDICES_TYPE, TYPE, DST_DIM, SRC_DIM, IDX_DIM>     \
    <<<smallIndexGrid, smallIndexBlock, 0, stream>>>(                                   \
      outInfo, selfInfo, indicesInfo,                                                   \
      outSelectDim, selfSelectDim, static_cast<TYPE>(sliceSize),                        \
      selfSelectDimSize);                                                               \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define LARGE_INDEX(TENSOR_TYPE, INDICES_TYPE, TYPE,                           \
                    DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR)                   \
  indexSelectLargeIndex<TENSOR_TYPE, INDICES_TYPE, TYPE,                       \
                        DST_DIM, SRC_DIM, IDX_DIM, IDX_IS_MAJOR>               \
    <<<largeIndexGrid, largeIndexBlock, 0, stream>>>(                          \
      outInfo, selfInfo, indicesInfo,                                          \
      outSelectDim, selfSelectDim, static_cast<TYPE>(outTotalSize),            \
      static_cast<TYPE>((IDX_IS_MAJOR) ? sliceSize : numIndices),              \
      selfSelectDimSize);                                                      \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  dim3 smallIndexGrid(std::min(ceil_div(sliceSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 smallIndexBlock(std::min(sliceSize, (ptrdiff_t)128));

  dim3 largeIndexGrid(std::min(ceil_div(outTotalSize, (ptrdiff_t)128), (ptrdiff_t)(mpc * 8)));
  dim3 largeIndexBlock(std::min(outTotalSize, (ptrdiff_t)128));
  if (cuda::detail::canUse32BitIndexMath(out) &&
      cuda::detail::canUse32BitIndexMath(self) &&
      cuda::detail::canUse32BitIndexMath(index)) {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, unsigned int>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto  selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, unsigned int>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);

    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<index_t, unsigned int>(index));
      indicesInfo.collapseDims();

      // A reasonable choice for when to have each thread iterate over
      // indices to choose
      if (numIndices <= 16) {
        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2);
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          SMALL_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2);
        } else {
          SMALL_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1);
        }
      } else {
        bool indexIsMajor = indexShouldBeMajor(outInfo, outSelectDim);

        if (outInfo.dims == 1 && selfInfo.dims == 1 && indContig) {
          LARGE_INDEX(scalar_t, index_t, unsigned int, 1, 1, -2, true);
        } else if (outInfo.dims == 2 && selfInfo.dims == 2 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 2, 2, -2, false);
          }
        } else if (outInfo.dims == 3 && selfInfo.dims == 3 && indContig) {
          if (indexIsMajor) {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, true);
          } else {
            LARGE_INDEX(scalar_t, index_t, unsigned int, 3, 3, -2, false);
          }
        } else {
          LARGE_INDEX(scalar_t, index_t, unsigned int, -1, -1, -1, true);
        }
      }
    });
  } else {
    auto outInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, uint64_t>(out));
    int outSelectDim = outInfo.collapseDims(dim);
    outInfo.reduceDim(outSelectDim);

    auto selfInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<scalar_t, uint64_t>(self));
    int selfSelectDim = selfInfo.collapseDims(dim);
    selfInfo.reduceDim(selfSelectDim);
    AT_DISPATCH_INDEX_TYPES(index.scalar_type(), "index_select_out_cuda_impl", [&] () {
      auto indicesInfo = tensorInfoLegacyIfScalar(cuda::detail::getTensorInfo<index_t, uint64_t>(index));
      indicesInfo.collapseDims();

      LARGE_INDEX(scalar_t, index_t, uint64_t, -1, -1, -1, true);
    });
  }
#undef SMALL_INDEX
#undef LARGE_INDEX
}
} // anonymous namespace

Tensor& index_select_out_cuda(
    const Tensor& self,
    int64_t dim,
    const Tensor& index,
    Tensor& out) {
  static constexpr string_view DIM_WARNING =
      "Tensor too large or too many (> 25) dimensions";
  TORCH_CHECK(
      at::cuda::check_device({out, self, index}),
      "Input, output and indices must be on the current device");
  at::assert_no_internal_overlap(out);
  at::assert_no_overlap(out, self);
  at::assert_no_overlap(out, index);

  dim = at::maybe_wrap_dim(dim, self);
  TORCH_CHECK(self.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);
  TORCH_CHECK(index.dim() <= MAX_TENSORINFO_DIMS, DIM_WARNING);
  if (self.is_quantized()){
    TORCH_CHECK(
      self.qscheme() == kPerTensorAffine,
      "Only per_tensor quantized quantized tensors are supported by index_select.")
    AT_DISPATCH_QINT_TYPES(out.scalar_type(), "index_select_quant_cuda", [&] {
      index_select_out_cuda_impl<scalar_t>(out, self, dim, index);
    });
  } else {
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
        at::ScalarType::Half,
        at::ScalarType::Bool,
        at::ScalarType::BFloat16,
        out.scalar_type(),
        "index_select_cuda",
        [&] { index_select_out_cuda_impl<scalar_t>(out, self, dim, index); });
  }

  return out;
}

Tensor index_select_cuda(const Tensor& self, int64_t dim, const Tensor& index) {
  Tensor out;
  if (self.is_quantized()){
    TORCH_CHECK(
      self.qscheme() == kPerTensorAffine,
      "Only per_tensor quantized quantized tensors are supported by index_select.")
    out = at::empty_quantized({0}, self);
  } else {
    out = at::empty({0}, self.options());
  }
  at::native::index_select_out_cuda(self, dim, index, out);
  return out;
}

namespace {

template <typename mask_t>
void masked_fill_kernel(TensorIterator& iter, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kBool, kHalf, kBFloat16, iter.common_dtype(), "masked_fill_", [&]() {
        const auto value_ = value.to<scalar_t>();
        gpu_kernel(
            iter, [value_] GPU_LAMBDA(scalar_t self, mask_t mask) -> scalar_t {
              if (mask) {
                return value_;
              }
              return self;
            });
      });
}

} // anonymous namespace

Tensor & masked_fill__cuda(Tensor& self, const Tensor & mask, const Scalar& value) {
  TORCH_CHECK(self.device() == mask.device(), "expected self and mask to be on the same device, but got mask on ",
    mask.device(), " and self on ", self.device());
  TORCH_CHECK(mask.scalar_type() == kByte || mask.scalar_type() == kBool,
    "expected mask dtype to be Bool but got ", mask.scalar_type());
  auto maybe_outnames = namedinference::broadcast_to_outnames(self, mask, "masked_fill_");
  if (at::has_internal_overlap(self) == MemOverlap::YES) {
    TORCH_WARN(
      "Use of masked_fill_ on expanded tensors is deprecated. "
      "Please clone() the tensor before performing this operation. "
      "This also applies to advanced indexing e.g. tensor[mask] = scalar");
  }
  at::assert_no_partial_overlap(self, mask);

  c10::MaybeOwned<Tensor> b_mask = expand_inplace(self, mask, "masked_fill_");

  auto iter = TensorIteratorConfig()
      .set_check_mem_overlap(false)
      .check_all_same_dtype(false)
      .resize_outputs(false)
      .add_output(self)
      .add_input(self)
      .add_input(*b_mask)
      .build();

  if (b_mask->dtype() == at::ScalarType::Byte) {
    TORCH_WARN("masked_fill_ received a mask with dtype torch.uint8, this behavior is now deprecated," \
            "please use a mask with dtype torch.bool instead.");
    masked_fill_kernel<uint8_t>(iter, value);
  } else {
    masked_fill_kernel<bool>(iter, value);
  }
  namedinference::propagate_names_if_nonempty(self, maybe_outnames);
  return self;
}

Tensor & masked_fill__cuda(Tensor& self, const Tensor & mask, const Tensor & value) {
  TORCH_CHECK(value.dim() == 0, "masked_fill_ only supports a 0-dimensional value tensor, but got tensor "
      "with ", value.dim(), " dimension(s).");
  return masked_fill__cuda(self, mask, value.item());
}

} // native
} // at
