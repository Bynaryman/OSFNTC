#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/cuda/CUDAGuard.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/TypeSafeSignMath.h>

#include <type_traits>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

template<typename scalar_t>
struct DivFunctor {
  __device__ scalar_t operator() (scalar_t a, scalar_t b) const {
    return a / b;
  }
};

template<typename T>
struct MulFunctor {
  __device__ T operator() (T a, T b) const {
    return a * b;
  }
};

// Workaround for the error: '*' in boolean context, suggest '&&' instead [-Werror=int-in-bool-context]
template<>
struct MulFunctor<bool> {
  __device__ bool operator() (bool a, bool b) const {
    return a && b;
  }
};


void div_true_kernel_cuda(TensorIteratorBase& iter) {
  if (iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_true_cuda", [&]() {
      using opmath_t = at::opmath_type<scalar_t>;
      auto inv_b = opmath_t(1.0) / iter.scalar_value<opmath_t>(2);
      iter.remove_operand(2);
      gpu_kernel(iter, BUnaryFunctor<scalar_t, scalar_t, scalar_t, MulFunctor<opmath_t>>(
        MulFunctor<opmath_t>(), inv_b));
    });
  } else {
    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "div_true_cuda", [&]() {
      DivFunctor<scalar_t> f;
      gpu_kernel_with_scalars(iter, f);
    });
  }
}

void div_trunc_kernel_cuda(TensorIteratorBase& iter) {
  auto dtype = iter.common_dtype();
  if (isIntegralType(dtype, /*includeBool*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(dtype, "div_trunc_cuda", [&]() {
      gpu_kernel_with_scalars(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        return a / b;
      });
    });
  } else if (iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, dtype, "div_trunc_cuda", [&]() {
      using accscalar_t = at::acc_type<scalar_t, true>;
      auto inv_b = accscalar_t(1.0) / iter.scalar_value<accscalar_t>(2);
      iter.remove_operand(2);
      gpu_kernel(iter, [inv_b] GPU_LAMBDA (scalar_t a) -> scalar_t {
        return std::trunc(a * inv_b);
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, dtype, "div_trunc_cuda", [&]() {
      gpu_kernel_with_scalars(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        return std::trunc(a / b);
      });
    });
  }
}

void div_floor_kernel_cuda(TensorIteratorBase& iter) {
  // See NOTE: [Floor Division in Python]
  const auto dtype = iter.common_dtype();
  if (dtype == kByte) {
    // In the special case of unsigned integer division, floor division is
    // equivalent to truncation division (since the signs of the divisor and
    // dividend are always the same)
    return div_trunc_kernel_cuda(iter);
  } else if (isIntegralType(dtype, /*includeBool*/ false)) {
    AT_DISPATCH_INTEGRAL_TYPES(dtype, "div_floor_cuda", [&]() {
      gpu_kernel_with_scalars(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        if (c10::signs_differ(a, b)) {
          // Subtracts one from the results of truncation division if the
          // divisor and dividend have different sign(bit)s and the remainder of
          // the division is nonzero
          const auto quot = a / b;
          const auto rem = a % b;
          return rem ? quot - 1 : quot;
        }

        return a / b;
      });
    });
  } else if (iter.is_cpu_scalar(2)) {
    // optimization for floating-point types: if the second operand is a CPU
    // scalar, compute a * reciprocal(b). Note that this may lose one bit of
    // precision compared to computing the division.
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, dtype, "div_floor_cuda", [&]() {
      using accscalar_t = at::acc_type<scalar_t, true>;
      auto b = iter.scalar_value<accscalar_t>(2);
      if (C10_UNLIKELY(b == 0)) {
        return div_true_kernel_cuda(iter);
      }

      auto inv_b = accscalar_t(1.0) / b;
      iter.remove_operand(2);
      gpu_kernel(iter, [b, inv_b] GPU_LAMBDA (scalar_t a) -> scalar_t {
        auto mod = std::fmod(a, b);
        auto div = (a - mod) * inv_b;
        if ((mod != 0) && (b < 0) != (mod < 0)) {
          div -= scalar_t(1);
        }

        scalar_t floordiv;
        if (div != 0) {
          floordiv = std::floor(div);
          if (div - floordiv > scalar_t(0.5)) {
            floordiv += scalar_t(1.0);
          }
        } else {
          floordiv = c10::cuda::compat::copysign(scalar_t(0), a * inv_b);
        }
        return floordiv;
      });
    });
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, dtype, "div_floor_cuda", [&]() {
      gpu_kernel_with_scalars(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
        if (C10_UNLIKELY(b == 0)) {
          return a / b;
        }

        auto mod = std::fmod(a, b);
        auto div = (a - mod) / b;
        if ((mod != 0) && (b < 0) != (mod < 0)) {
          div -= scalar_t(1);
        }

        scalar_t floordiv;
        if (div != 0) {
          floordiv = std::floor(div);
          if (div - floordiv > scalar_t(0.5)) {
            floordiv += scalar_t(1.0);
          }
        } else {
          floordiv = c10::cuda::compat::copysign(scalar_t(0), a / b);
        }
        return floordiv;
      });
    });
  }
}

void mul_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(kHalf, kBFloat16, kBool, iter.common_dtype(), "mul_cuda", [&]() {
    using opmath_t = at::opmath_type<scalar_t>;
    opmath_gpu_kernel_with_scalars<scalar_t>(iter, MulFunctor<opmath_t>());
  });
}

REGISTER_DISPATCH(div_true_stub, &div_true_kernel_cuda);
REGISTER_DISPATCH(div_trunc_stub, &div_trunc_kernel_cuda);
REGISTER_DISPATCH(div_floor_stub, &div_floor_kernel_cuda);
REGISTER_DISPATCH(mul_stub, &mul_kernel_cuda);

}} // namespace at::native
