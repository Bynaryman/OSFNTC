#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/cuda/jit_utils.h>
#include <ATen/NumericUtils.h>
#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/util/complex.h>

namespace at {
namespace native {

void exp2_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "exp2_cuda",
      [&]() {
        gpu_kernel(iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::exp2(a);
        });
      });
}

void i0_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "i0_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      using opmath_t = at::opmath_type<scalar_t>;
      // implicit conversion of a to opmath_t will happen here,
      //   but as far as TI is concerned, it's still a no-dynamic-cast kernel because lambda input is scalar_t
      return calc_i0<opmath_t>(a);
    });
  });
}

void i0e_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "i0e_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      using opmath_t = at::opmath_type<scalar_t>;
      return calc_i0e<opmath_t>(a);
    });
  });
}

// See note [Jiterator]
const char i1_name[] = "i1";
void i1_kernel_cuda(TensorIteratorBase& iter) {
  #ifdef USE_JITERATOR
    AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "i1_cuda", [&]() {
      jitted_gpu_kernel</*name=*/i1_name,
                        /*return_dtype=*/ scalar_t,
                        /*common_dtype=*/ scalar_t,
                        /*arity=*/ 1>(iter, i1_string);
    });
  #else
    AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "i1_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_i1(a);
      });
    });
  #endif // USE_JITERATOR
}

void i1e_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "i1e_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_i1e(a);
    });
  });
}

void sigmoid_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "sigmoid_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return static_cast<scalar_t>(1) / (static_cast<scalar_t>(1) + std::exp(-a));
    });
  });
}

void sinc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "sinc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          if (a == scalar_t(0)) {
            return scalar_t(1);
          } else {
            // NVCC says constexpr var is not accessible from device
            scalar_t product = c10::detail::pi<scalar_t>() * a;
            return std::sin(product) / product;
          }
        });
      });
}

void logit_kernel_cuda(TensorIteratorBase& iter, const Scalar& eps_scalar) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.common_dtype(),
      "logit_cuda",
      [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        const T_ACC eps = eps_scalar.to<T_ACC>();
        if (eps < T_ACC(0)) {
          gpu_kernel(iter, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
            const T_ACC x_acc = static_cast<T_ACC>(x);
            return c10::cuda::compat::log(x_acc / (T_ACC(1) - x_acc));
          });
        } else {
          const T_ACC lo = eps;
          const T_ACC hi = T_ACC(1) - eps;
          gpu_kernel(
              iter, [lo, hi] GPU_LAMBDA(scalar_t x) -> scalar_t {
                const T_ACC x_acc = static_cast<T_ACC>(x);
                T_ACC z = x_acc < lo ? lo : (x_acc > hi ? hi : x_acc);
                return c10::cuda::compat::log(z / (T_ACC(1) - z));
              });
        }
      });
}

void ndtri_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "ndtri_cuda", [&]() {
    gpu_kernel(
        iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return calc_ndtri(a); });
  });
}

void erf_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "erf_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erf(a);
    });
  });
}

void erfc_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half, ScalarType::BFloat16,
      iter.common_dtype(), "erfc_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::erfc(a);
        });
      });
}

void erfinv_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "erfinv_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::erfinv(a);
    });
  });
}

void erfcx_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES(iter.common_dtype(), "erfcx_cuda", [&]() {
    gpu_kernel(
        iter, [] GPU_LAMBDA(scalar_t a) -> scalar_t { return calc_erfcx(a); });
  });
}

void kaiser_window_kernel_cuda(TensorIteratorBase& iter, int64_t window_length, double beta_){
  AT_DISPATCH_FLOATING_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.dtype(), "kaiser_window_cuda", [&](){
    using opmath_t = at::opmath_type<scalar_t>;
    const opmath_t inv_alpha = static_cast<opmath_t>(2.0 / (window_length - 1));
    const opmath_t beta = static_cast<opmath_t>(beta_);
    const opmath_t inv_i0_beta = 1.0 / calc_i0(beta);
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t a) -> scalar_t {
      opmath_t x = static_cast<opmath_t>(a) * inv_alpha - 1;
      opmath_t y = std::max<opmath_t>(0, 1 - x * x);
      return calc_i0(beta * ::sqrt(y)) * inv_i0_beta;
    });
  });
}

void entr_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::Half,
      ScalarType::BFloat16,
      iter.common_dtype(),
      "entr_cuda",
      [&]() {
        gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t x) -> scalar_t {
          if (at::_isnan(x)) {
            return x;
          } else if (x > 0) {
            return -x * std::log(x);
          } else if (x == 0) {
            return 0;
          }
          return static_cast<scalar_t>(-INFINITY);
        });
      });
}

REGISTER_DISPATCH(exp2_stub, &exp2_kernel_cuda);
REGISTER_DISPATCH(i0_stub, &i0_kernel_cuda);
REGISTER_DISPATCH(special_i0e_stub, &i0e_kernel_cuda);
REGISTER_DISPATCH(special_i1_stub, &i1_kernel_cuda);
REGISTER_DISPATCH(special_i1e_stub, &i1e_kernel_cuda);
REGISTER_DISPATCH(sigmoid_stub, &sigmoid_kernel_cuda);
REGISTER_DISPATCH(sinc_stub, &sinc_kernel_cuda);
REGISTER_DISPATCH(logit_stub, &logit_kernel_cuda);
REGISTER_DISPATCH(erf_stub, &erf_kernel_cuda);
REGISTER_DISPATCH(erfc_stub, &erfc_kernel_cuda);
REGISTER_DISPATCH(erfinv_stub, &erfinv_kernel_cuda);
REGISTER_DISPATCH(kaiser_window_stub, &kaiser_window_kernel_cuda);
REGISTER_DISPATCH(special_entr_stub, &entr_kernel_cuda);
REGISTER_DISPATCH(special_ndtri_stub, &ndtri_kernel_cuda);
REGISTER_DISPATCH(special_erfcx_stub, &erfcx_kernel_cuda);

} // namespace native
} // namespace at
