#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/AccumulateType.h>

// NOTE: CUDA on Windows requires that the enclosing function
// of a __device__ lambda not have internal linkage.

namespace at { namespace native {

void logaddexp_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(
      ScalarType::BFloat16,
      iter.dtype(), "logaddexp_cuda",
      [&]() {
        using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;
        gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
          if (::isinf(static_cast<accscalar_t>(a)) && a == b) {
            return a;
          }
          else {
            scalar_t m = ::max(a, b);
            return m + ::log((scalar_t)(1.0) + ::exp(-::abs(a - b)));
          }
        });
      });
}

void logaddexp2_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(
      ScalarType::BFloat16,
      iter.dtype(), "logaddexp2_cuda",
      [&]() {
        using accscalar_t = at::acc_type<scalar_t, /*is_cuda=*/true>;
        gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a, scalar_t b) -> scalar_t {
          if (::isinf(static_cast<accscalar_t>(a)) && a == b) {
            return a;
          }
          else {
            scalar_t m = ::max(a, b);
            return m + ::log2((scalar_t)(1.0) + ::pow((scalar_t)(2.0), -::abs(a - b)));
          }
        });
      });
}

REGISTER_DISPATCH(logaddexp_stub, &logaddexp_kernel_cuda);
REGISTER_DISPATCH(logaddexp2_stub, &logaddexp2_kernel_cuda);

}} // namespace at::native
