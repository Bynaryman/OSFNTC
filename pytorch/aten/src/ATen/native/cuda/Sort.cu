#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/Sort.h>
#include <ATen/core/TensorBase.h>
#include <ATen/core/Array.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/cub.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/KernelUtils.h>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/SortUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>

#include <limits>

namespace at { namespace native {

// In alignment with default sort on a c++ map, this function
// will permute key and value tensors identically, and
// in such a way that the 'key' tensor is ordered numerically
void sortKeyValueInplace(const TensorBase& key,
                         const TensorBase& value,
                         int dim, bool dir) {
  TORCH_CHECK(key.sizes() == value.sizes(),
              "Key tensor must have same size as value tensor");
  int dims = value.dim();
  TORCH_CHECK(dims <= MAX_DIMS, "value tensor has too many dimensions");
  // if key and value tensors have the same size, we do not need to check both

  ptrdiff_t inElements = key.numel();

  if (inElements == 0) {
    return;
  }

  int64_t keySliceSize = key.size(dim);
  ptrdiff_t keySlices = inElements / keySliceSize;

  // The amount of shared memory and block size is based on
  // 2^ceil(lg(n)); we choose that sorting implementation for a given
  // size.
  int64_t ceilPowerOf2 = nextHighestPowerOf2(keySliceSize);

  // FIXME: We'd have to find some other trick with Thrust to perform a
  // vectorized (key, value) sort by slice segment
  TORCH_INTERNAL_ASSERT(ceilPowerOf2 <= 2048, "sortKeyValueInplace only works for sizes <= 2048 at present");

  // The grid is based on the number of independent slices that we
  // have to sort; one block per slice
  dim3 grid;
  TORCH_INTERNAL_ASSERT(getGridFromTiles(keySlices, grid), "Too many slices to sort");

#define HANDLE_CASE(TYPE, A, SIZE)                                      \
  do {                                                                  \
    int blockSize = SIZE / 2;                                           \
    if (blockSize < 1) {                                                \
      blockSize = 1;                                                    \
    }                                                                   \
                                                                        \
    dim3 block(blockSize);                                              \
                                                                        \
    if (dir) {                                                          \
      bitonicSortKVInPlace<scalar_t, int64_t, A, -1,                    \
          GTOp<scalar_t, true>, TYPE, SIZE>                           \
        <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(        \
          keyInfo,                                                      \
          keySlices,                                                    \
          (TYPE) keySliceSize,                                          \
          (TYPE) keyInfo.strides[collapseKeyDim],                       \
          valueInfo,                                                    \
          (TYPE) valueInfo.strides[collapseValueDim],                   \
          GTOp<scalar_t, true>());                                    \
      C10_CUDA_KERNEL_LAUNCH_CHECK();                                   \
    } else {                                                            \
      bitonicSortKVInPlace<scalar_t, int64_t, A, -1,                    \
      LTOp<scalar_t, true>, TYPE, SIZE>                               \
        <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(        \
          keyInfo,                                                      \
          keySlices,                                                    \
          (TYPE) keySliceSize,                                          \
          (TYPE) keyInfo.strides[collapseKeyDim],                       \
          valueInfo,                                                    \
          (TYPE) valueInfo.strides[collapseValueDim],                   \
          LTOp<scalar_t, true>());                                    \
      C10_CUDA_KERNEL_LAUNCH_CHECK();                                   \
    }                                                                   \
  } while (0)

#define HANDLE_SORT_CASE(TYPE, A)                       \
  {                                                     \
    switch (ceilPowerOf2) {                             \
      case 2048:                                        \
      HANDLE_CASE(TYPE, A, 2048);                       \
      break;                                            \
      case 1024:                                        \
      case 512:                                         \
      case 256:                                         \
      HANDLE_CASE(TYPE, A, 1024);                       \
      break;                                            \
      case 128:                                         \
      case 64:                                          \
      HANDLE_CASE(TYPE, A, 128);                        \
      break;                                            \
      case 32:                                          \
      case 16:                                          \
      case 8:                                           \
      case 4:                                           \
      case 2:                                           \
      HANDLE_CASE(TYPE, A, 32);                         \
      break;                                            \
      case 1:                                           \
      /* Nothing to do, data already sorted */          \
      break;                                            \
      default:                                          \
      TORCH_INTERNAL_ASSERT(false);                     \
    }                                                   \
  }

  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  // The constructed key/value tensor info is used to select the slice
  // we are sorting on a per-block basis
  AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Half, at::ScalarType::BFloat16, at::ScalarType::Bool, key.scalar_type(), "sortKeyValueInplace", [&]  {
    if (at::cuda::detail::canUse32BitIndexMath(key)) {
      at::cuda::detail::TensorInfo<scalar_t, unsigned int> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, unsigned int>(key);
      at::cuda::detail::TensorInfo<int64_t, unsigned int> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, unsigned int>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      if (keyInfo.isContiguous()) {
        HANDLE_SORT_CASE(unsigned int, -2);
      } else {
        switch (keyInfo.dims) {
          case 2:
            HANDLE_SORT_CASE(unsigned int, 2);
            break;
          default:
            HANDLE_SORT_CASE(unsigned int, -1);
            break;
        }
      }

    } else {
      at::cuda::detail::TensorInfo<scalar_t, uint64_t> keyInfo =
        at::cuda::detail::getTensorInfo<scalar_t, uint64_t>(key);
      at::cuda::detail::TensorInfo<int64_t, uint64_t> valueInfo =
        at::cuda::detail::getTensorInfo<int64_t, uint64_t>(value);

      auto strideKey = keyInfo.strides[dim];
      keyInfo.sizes[dim] = 1;
      int collapseKeyDim = keyInfo.collapseDims(dim);
      keyInfo.strides[collapseKeyDim] = strideKey;
      auto strideValue = valueInfo.strides[dim];
      valueInfo.sizes[dim]=1;
      int collapseValueDim = valueInfo.collapseDims(dim);
      valueInfo.strides[collapseValueDim] = strideValue;

      // int64_t case is rare, just instantiate the generic version
      HANDLE_SORT_CASE(uint64_t, -1);
    }
  });
#undef HANDLE_CASE
#undef HANDLE_SORT_CASE
#undef HANDLE_A_CASE
}

namespace {

struct offset_t {
  int stride;
  int begin;
  __device__ int operator[](int i) {
    return stride * (begin + i);
  }
};

}

namespace {

// Segmented sort by full sort algorithm:.
// Say we are sorting a (2, 3) tensor. We have in flattened form:
// values       0.4 1.2 5.3 6.2 1.3 2.3
// indices        0   1   2   0   1   2
// segment_id     0   0   0   1   1   1

// First we sort by values, globally:
// values       6.2 5.3 2.3 1.2 1.3 0.4
// indices        0   2   2   1   1   0
// segment_id     1   0   1   0   1   0

// Then we stable sort by segment id:
// values       5.3 1.2 0.4 6.2 2.3 1.3
// indices        2   1   0   0   2   1
// segment_id     0   0   0   1   1   1

// This method can only work if the slice we are sorting (`dim`) is
// innermost, and both values and indices are contiguous. We do this
// by re-arranging the input into this form as needed, which will
// unfortunately allocate memory if the request is not in this form.
// Vectorized sort is slower than iterated sort if the number of
// slices is small (since we're sorting twice, instead of invoking a
// smaller sort `numSlices` times), but the cub sort
// implementation here is a catch-all, so we're not looking for
// efficiency, but instead correctness.

template<typename scalar_t>
__global__ void sort_postprocess_kernel(const scalar_t *in, scalar_t *out, int64_t *index, const int2 *i_s_ptr, int nsegments, int nsort) {
  CUDA_KERNEL_LOOP(i, nsegments * nsort) {
    int segment = i / nsort;
    int j = i % nsort;

    int offset = segment * nsort;
    const scalar_t *in_ = in + offset;
    scalar_t *out_ = out + offset;
    int64_t *index_ = index + offset;
    const int2 *i_s_ptr_ = i_s_ptr + offset;

    int idx = i_s_ptr_[j].y;
    index_[j] = idx;
    out_[j] = in_[idx];
  }
}


__global__ void fill_index_and_segment_kernel(
    int2 *data, int numel, at::cuda::detail::IntDivider<uint32_t> nsort_divider) {
  CUDA_KERNEL_LOOP(idx, numel) {
    auto div_mod = nsort_divider.divmod(idx);
    auto segment = static_cast<int>(div_mod.div);
    auto sort = static_cast<int>(div_mod.mod);
    data[idx] = int2{segment, sort};
  }
}

__global__ void fill_reverse_indices_kernel(
    int64_t *data, int numel, at::cuda::detail::IntDivider<uint32_t> nsort_divider) {
  CUDA_KERNEL_LOOP(idx, numel) {
    data[idx] = nsort_divider.mod(idx);
  }
}


template<typename scalar_t>
inline void segmented_sort_pairs_by_full_sort(
  int64_t nsegments, int64_t nsort, int64_t n, bool descending, const TensorBase &indices,
  const scalar_t *self_ptr, scalar_t *values_ptr, int64_t *indices_ptr
) {
  int64_t segment_bits = std::max<int64_t>(1L, static_cast<int64_t>(std::ceil(std::log2(nsegments))));

  const auto numel = nsort * nsegments;
  auto cuda_allocator = at::cuda::getCUDADeviceAllocator();
  auto indices_and_segment = cuda_allocator->allocate(numel * sizeof(int2));
  auto i_s_ptr = static_cast<int2 *>(indices_and_segment.get());

  using namespace at::cuda::detail;
  dim3 block = CUDA_NUM_THREADS;
  dim3 grid = GET_BLOCKS(numel);
  auto stream = c10::cuda::getCurrentCUDAStream();
  at::cuda::detail::IntDivider<uint32_t> nsort_divider(nsort);
  fill_index_and_segment_kernel<<<grid, block, 0, stream>>>(
      i_s_ptr, numel, nsort_divider);

  auto indices_and_segment2 = cuda_allocator->allocate(nsegments * nsort * sizeof(int2));
  auto i_s_ptr2 = static_cast<int2 *>(indices_and_segment2.get());

  at::cuda::hipcub::radix_sort_pairs<scalar_t, int2>(
    self_ptr, nullptr, i_s_ptr, i_s_ptr2,
    n, descending);

  TORCH_INTERNAL_ASSERT(segment_bits <= 32);

  // sort on lower 32bits, i.e. segment index
  at::cuda::hipcub::radix_sort_keys<int64_t>(
    reinterpret_cast<int64_t *>(i_s_ptr2), reinterpret_cast<int64_t *>(i_s_ptr),
    n, false, 0, segment_bits);

  sort_postprocess_kernel<<<(n + 511) / 512, 512, 0, at::cuda::getCurrentCUDAStream()>>>(
    self_ptr, values_ptr, indices_ptr, i_s_ptr, nsegments, nsort);
}

template<typename scalar_t>
void segmented_sort_pairs(
    int64_t nsegments, int64_t nsort, int64_t n, bool descending,
    const scalar_t *self_ptr, scalar_t *values_ptr, int64_t *indices_ptr) {
  const auto numel = nsort * nsegments;
  auto cuda_allocator = at::cuda::getCUDADeviceAllocator();
  auto reverse_indices = cuda_allocator->allocate(numel * sizeof(int64_t));
  int64_t *reverse_indices_ptr = static_cast<int64_t *>(reverse_indices.get());

  using namespace at::cuda::detail;
  dim3 block = CUDA_NUM_THREADS;
  dim3 grid = GET_BLOCKS(numel);
  auto stream = c10::cuda::getCurrentCUDAStream();
  at::cuda::detail::IntDivider<uint32_t> nsort_divider(nsort);
  fill_reverse_indices_kernel<<<grid, block, 0, stream>>>(
      reverse_indices_ptr, numel, nsort_divider);

  at::cuda::hipcub::segmented_sort_pairs(self_ptr, values_ptr,
                                      reverse_indices_ptr, indices_ptr, n, nsegments,
                                      offset_t{(int)nsort, 0}, offset_t{(int)nsort, 1}, descending);
}

}  // namespace

void launch_stable_sort_kernel(
    const TensorBase &self, int64_t dim, bool descending,
    const TensorBase &values, const TensorBase &indices) {
  const auto numel = self.numel();
  if (numel == 0) {
    return;
  }

  int64_t numel_or_intmax = std::min(numel, static_cast<int64_t>(std::numeric_limits<int>::max()));
  int64_t nsort = self.size(dim);
  int64_t nbatch = (numel_or_intmax / nsort) * nsort;
  TORCH_CHECK(nbatch > 0, "Cannot sort dimension of length ", nsort);
  int64_t *indices_ptr = indices.data_ptr<int64_t>();

#if defined(USE_ROCM)
  constexpr bool is_rocm = true;
#else
  constexpr bool is_rocm = false;
#endif

  AT_DISPATCH_ALL_TYPES_AND3(kBool, kHalf, kBFloat16, self.scalar_type(), "sort", [&]{
    c10::guts::if_constexpr<!(is_rocm && std::is_same<scalar_t, c10::BFloat16>::value)>([&](auto _){
      const scalar_t *self_ptr = self.data_ptr<scalar_t>();
      scalar_t *values_ptr = values.data_ptr<scalar_t>();
      int64_t remaining = _(numel);
      while (remaining > 0) {
        int64_t n = std::min(remaining, nbatch);
        int64_t nsegments = n / nsort;

        if (nsegments < 128) {
          segmented_sort_pairs_by_full_sort(nsegments, nsort, n, descending,
            indices, self_ptr, values_ptr, indices_ptr);
        } else {
          segmented_sort_pairs(nsegments, nsort, n, descending,
                               self_ptr, values_ptr, indices_ptr);
        }

        remaining -= n;
        self_ptr += n;
        values_ptr += n;
        indices_ptr += n;
      }
    }, [&](auto _){ TORCH_CHECK(_(false), "BFloat16 is not supported on ROCm"); });
  });
}

}}  // namespace at::native
