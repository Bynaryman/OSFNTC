#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <ATen/native/Activation.h>

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/core/Scalar.h>

namespace at {
namespace native {

// -----------------------------------
// glu forward
// -----------------------------------
void glu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "glu_cuda", [&]() {
    using acc_t = at::acc_type<scalar_t, true>;
    gpu_kernel(iter, [] GPU_LAMBDA (scalar_t a_, scalar_t b_) -> scalar_t {
      const acc_t a = a_;
      const acc_t b = b_;
      const acc_t one = acc_t(1);
      const acc_t sigmoid = one / (one + std::exp(-b));
      return a * sigmoid;
    });
  });
}

// -----------------------------------
// glu backward
// -----------------------------------

// Byte offsets don't require multiplication by sizeof(T), so are slightly cheaper.
// For fixed offsets, this removes all penalty from 64-bit indexing.
template <typename T>
__device__ T* byte_offset(T* ptr, int64_t offset) {
  using byte_ptr_t = typename std::conditional<
    std::is_const<T>::value, const char*, char*>::type;
  return reinterpret_cast<T*>(
    reinterpret_cast<byte_ptr_t>(ptr) + offset
  );
}

template <typename scalar_t, typename OffsetCalc>
__global__ void glu_backward_kernel(
    int numel, scalar_t* gI, const scalar_t* I, const scalar_t* gO,
    OffsetCalc offset_calculator,
    int64_t gI_byte_offset, int64_t I_byte_offset) {
  using acc_t = at::acc_type<scalar_t, true>;

  const uint32_t linear_index = blockIdx.x * blockDim.x + threadIdx.x;
  if (linear_index >= numel) {
    return;
  }
  const auto offsets = offset_calculator.get(linear_index);

  // We explicitly iterate over the first half of the input tensor, and
  // gI_byte_offset and I_byte_offset are the offsets to access the
  // corresponding index in the second half of the tensor.
  const acc_t a = I[offsets[1]];
  const acc_t b = *byte_offset(I + offsets[1], I_byte_offset);
  const acc_t gO_val = gO[offsets[2]];

  const auto one = acc_t(1);
  const acc_t sigmoid = one / (one + std::exp(-b));

  auto* gA = gI + offsets[0];
  *gA = sigmoid * gO_val;

  auto* gB = byte_offset(gA, gI_byte_offset);
  *gB = (one - sigmoid) * sigmoid * gO_val * a;
}

void launch_glu_backward_kernel(const TensorIteratorBase& iter,
                                int64_t gI_stride, int64_t I_stride) {
  const auto N = iter.numel();
  TORCH_INTERNAL_ASSERT_DEBUG_ONLY(N > 0 && N <= std::numeric_limits<int32_t>::max());
  const auto offset_calculator = make_element_offset_calculator<3>(iter);
  constexpr int64_t block_size = 256;
  const int64_t grid = (N + block_size - 1) / block_size;
  const auto stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "glu_backward_cuda", [&] {
    auto gI = static_cast<scalar_t*>(iter.data_ptr(0));
    auto I = static_cast<const scalar_t*>(iter.data_ptr(1));
    auto gO = static_cast<const scalar_t*>(iter.data_ptr(2));
    glu_backward_kernel<<<grid, block_size, 0, stream>>>(
        N, gI, I, gO, offset_calculator,
        gI_stride * sizeof(scalar_t), I_stride * sizeof(scalar_t));
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

// -----------------------------------
// log_sigmoid forward
// -----------------------------------

void launch_log_sigmoid_forward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(kHalf, iter.common_dtype(),
                                 "log_sigmoid_forward_cuda", [&] {
    using acc_t = acc_type<scalar_t, true>;
    gpu_kernel(iter,
        [] GPU_LAMBDA (scalar_t in_) -> scalar_t {
          const acc_t in = in_;
          const auto min = std::min(acc_t(0), in);
          const auto z = std::exp(-std::abs(in));
          return min - std::log1p(z);
        });
  });
}

// -----------------------------------
// log_sigmoid backward
// -----------------------------------

void log_sigmoid_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(kHalf, iter.common_dtype(),
                                 "log_sigmoid_backward_cuda", [&] {
    using acc_t = acc_type<scalar_t, true>;
    gpu_kernel(iter,
        [] GPU_LAMBDA (scalar_t in_, scalar_t grad_out_) -> scalar_t {
          const acc_t in = in_;
          const acc_t grad_out = grad_out_;

          auto in_negative = in < acc_t(0);
          auto max_deriv = in_negative ? acc_t(1) : acc_t(0);
          auto sign = in_negative ? acc_t(1) : -acc_t(1);
          const auto z = std::exp(-std::abs(in));
          return grad_out * (max_deriv - sign * (z / (acc_t(1) + z)));
        });
  });
}

// -----------------------------------
// prelu forward
// -----------------------------------
void launch_prelu_cuda_kernel_share_weights(TensorIteratorBase &iter, const TensorBase &weight) {
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.input_dtype(), "prelu_cuda", [&] {
    const auto *weight_data = weight.data_ptr<scalar_t>();
    at::native::gpu_kernel(iter,
        [weight_data] GPU_LAMBDA (scalar_t input_val) {
          return (input_val > 0) ? input_val : *weight_data * input_val;
        });
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_kernel_multi_weights(
  scalar_t* result_data,
  const scalar_t* input_data,
  const scalar_t* weight_data,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;

  // multiply values at each channel with weight[channel_index]
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  result_data[linearId] = (input_data_val > 0) ? input_data_val : weight_data[channel] * input_data_val;
}

void launch_prelu_cuda_kernel_multi_weights(
    const TensorBase &result, const TensorBase &input, const TensorBase &weight) {
  int64_t input_ndim = input.dim();
  TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

  int64_t channel_size = 1; // channel_size default to 1
  int64_t input_stride0 = 1, input_stride1 = 1;

  if (input_ndim > 1) {
    channel_size = input.size(1); // channel is the 2nd dim of input
    auto strides = input.strides();
    input_stride0 = strides[0];
    input_stride1 = strides[1];
  }
  const int64_t weight_num = weight.numel();
  TORCH_CHECK(channel_size == weight_num,
    "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
    " and channel size = ", channel_size, ".");

  // config to run cuda kernel
  int64_t input_numel = input.numel();
  const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
  dim3 grid;
  int curDevice = -1;
  hipGetDevice(&curDevice);
  hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
  TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu: input too large or too many dimensions");

  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_cuda", [&] {
    prelu_cuda_kernel_multi_weights<scalar_t>
    <<<grid, block, 0, stream>>>(
      result.data_ptr<scalar_t>(),
      input.data_ptr<scalar_t>(),
      weight.data_ptr<scalar_t>(),
      input_stride0,
      input_stride1,
      input_numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

// -----------------------------------
// prelu backward
// -----------------------------------
void launch_prelu_cuda_backward_kernel_share_weights(
    TensorIteratorBase &iter, const TensorBase &weight) {
  // N.B. `std::tuple` does not support `::operator=` on device code.
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.input_dtype(), "prelu_backward_cuda", [&] {
    const auto *weight_data = weight.data_ptr<scalar_t>();
    gpu_kernel_multiple_outputs(iter, [=] GPU_LAMBDA (scalar_t input, scalar_t grad_out) -> thrust::tuple<scalar_t, scalar_t> {
        scalar_t input_grad = input > 0 ? grad_out : (*weight_data) * grad_out;
        scalar_t weight_grad_collector = input > 0 ? scalar_t(0) : input * grad_out;
        return {input_grad, weight_grad_collector};
      });
  });
}

template <typename scalar_t>
__global__ void prelu_cuda_backward_kernel_multi_weights(
  const scalar_t* input_data,
  const scalar_t* weight_data,
  const scalar_t* grad_out_data,
  scalar_t* input_grad_data,
  scalar_t* weight_grad_collector,
  int64_t input_stride0,
  int64_t input_stride1,
  int64_t input_numel) {

  int64_t linearId = blockIdx.x * blockDim.x + threadIdx.x;
  if (linearId >= input_numel) return;
  int64_t channel = (linearId % input_stride0) / input_stride1;
  scalar_t input_data_val = input_data[linearId];
  scalar_t grad_out_data_val = grad_out_data[linearId];
  input_grad_data[linearId] = (input_data_val > 0) ? grad_out_data_val : weight_data[channel] * grad_out_data_val;
  weight_grad_collector[linearId] = (input_data_val > 0) ? scalar_t(0) : input_data_val * grad_out_data_val;
}

void launch_prelu_cuda_backward_kernel_multi_weights(
    const TensorBase &input, const TensorBase &weight, const TensorBase &grad_out,
    const TensorBase &input_grad, const TensorBase &weight_grad_collector) {
  int64_t input_ndim = input.dim();
  TORCH_CHECK(input_ndim > 0, "Not allow zero-dim input tensor.");

  int64_t channel_size = 1; // channel_size default to 1
  int64_t input_stride0 = 1, input_stride1 = 1;

  if (input_ndim > 1) {
    channel_size = input.size(1); // channel is the 2nd dim of input
    auto strides = input.strides();
    input_stride0 = strides[0];
    input_stride1 = strides[1];
  }
  const int64_t weight_num = weight.numel();
  TORCH_CHECK(channel_size == weight_num,
    "Mismatch of parameter numbers and input channel size. Found parameter numbers = ", weight_num,
    " and channel size = ", channel_size, ".");

  // config to run cuda kernel
  int64_t input_numel = input.numel();
  const dim3 block = dim3(std::min(static_cast<int64_t>(cuda::getApplyBlock().x), input_numel));
  dim3 grid;
  int curDevice = -1;
  hipGetDevice(&curDevice);
  hipStream_t stream = at::cuda::getCurrentCUDAStream(curDevice);
  TORCH_CHECK(cuda::getApplyGrid(input_numel, grid, curDevice), "prelu_backward_cuda: input too large or too many dimensions");

  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, input.scalar_type(), "prelu_backward_cuda", [&] {
    prelu_cuda_backward_kernel_multi_weights<scalar_t>
    <<<grid, block, 0, stream>>>(
      input.data_ptr<scalar_t>(),
      weight.data_ptr<scalar_t>(),
      grad_out.data_ptr<scalar_t>(),
      input_grad.data_ptr<scalar_t>(),
      weight_grad_collector.data_ptr<scalar_t>(),
      input_stride0,
      input_stride1,
      input_numel);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  });
}

// -----------------------------------
// hardshrink
// -----------------------------------
void hardshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a >= -lambd && a <= lambd) ? scalar_t(0) : a;
    });
  });
}

void softshrink_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softshrink_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > lambd ? a - lambd : (a < -lambd ? a + lambd : scalar_t(0));
    });
  });
}

void shrink_backward_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "shrink_backward_cuda", [&]() {
    auto lambd = value.to<scalar_t>();
    gpu_kernel(iter, [lambd]GPU_LAMBDA(scalar_t grad_val, scalar_t self_val) -> scalar_t {
      return (self_val >= -lambd && self_val <= lambd) ? scalar_t(0) : grad_val;
    });
  });
}

void hardtanh_backward_kernel(TensorIterator& iter, const Scalar& min, const Scalar& max) {
  AT_DISPATCH_FLOATING_TYPES_AND(at::ScalarType::Half, iter.dtype(), "hardtanh_backward_cuda", [&]() {
    auto min_val = min.to<scalar_t>();
    auto max_val = max.to<scalar_t>();
    gpu_kernel(iter, [min_val, max_val]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return (b <= min_val) || (b >= max_val) ? scalar_t(0) : a;
    });
  });
}

void softplus_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (a * beta) > threshold ? a : static_cast<scalar_t>(::log1p(std::exp(a * beta))) / beta;
    });
  });
}

void softplus_backward_kernel(TensorIteratorBase& iter, const Scalar& beta_, const Scalar& threshold_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "softplus_backward_cuda", [&]() {
    auto beta = beta_.to<scalar_t>();
    auto threshold = threshold_.to<scalar_t>();
    gpu_kernel(iter, [beta, threshold]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      scalar_t z = std::exp(b * beta);
      return (b * beta) > threshold ? a : a * z / (z + scalar_t(1.));
    });
  });
}

template <typename scalar_t>
void threshold_kernel_impl(TensorIteratorBase& iter, scalar_t threshold, scalar_t value) {
  gpu_kernel_with_scalars(iter, [=]GPU_LAMBDA(scalar_t x, scalar_t other) -> scalar_t {
    return x <= threshold ? value : other;
  });
}

static void threshold_kernel_cuda(TensorIteratorBase& iter, const Scalar& threshold, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "threshold_cuda", [&] {
    threshold_kernel_impl<scalar_t>(iter, threshold.to<scalar_t>(), value.to<scalar_t>());
  });
}

void elu_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a * poscoef : (static_cast<scalar_t>(std::exp(a * negiptcoef)) - scalar_t(1.)) * negcoef;
    });
  });
}

void elu_backward_kernel(TensorIteratorBase& iter, const Scalar& alpha, const Scalar& scale, const Scalar& input_scale, bool is_result) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "elu_backward_cuda", [&]() {
    auto negcoef = alpha.to<scalar_t>() * scale.to<scalar_t>();
    auto poscoef = scale.to<scalar_t>();
    auto negiptcoef = input_scale.to<scalar_t>();
    gpu_kernel(iter, [negcoef, poscoef, negiptcoef, is_result]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      if (is_result) {
        return b <= scalar_t(0) ? a * negiptcoef * (b + negcoef) : a * poscoef;
      } else {
        return b <= scalar_t(0) ? a * negiptcoef * negcoef * (static_cast<scalar_t>(std::exp(b * negiptcoef))) : a * poscoef;
      }
    });
  });
}

void GeluCUDAKernelImpl(TensorIteratorBase& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, it.dtype(), "GeluCUDAKernelImpl", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    gpu_kernel(it, [] GPU_LAMBDA(scalar_t x) -> scalar_t {
      return static_cast<T_ACC>(x) *
          c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
    });
  });
}

void GeluBackwardCUDAKernelImpl(TensorIteratorBase& it) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16,
      it.dtype(), "GeluBackwardCUDAKernelImpl", [&]() {
        using T_ACC = acc_type<scalar_t, true>;
        gpu_kernel(it, [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          constexpr T_ACC kBeta = M_2_SQRTPI * M_SQRT1_2 * T_ACC(0.5);
          const T_ACC cdf = c10::cuda::compat::normcdf(static_cast<T_ACC>(x));
          const T_ACC pdf =
              c10::cuda::compat::exp(
                  T_ACC(-0.5) * static_cast<T_ACC>(x) * static_cast<T_ACC>(x)) *
              kBeta;
          return static_cast<T_ACC>(dy) * (cdf + static_cast<T_ACC>(x) * pdf);
        });
      });
}

namespace {

void leaky_relu_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a) -> scalar_t {
      return a > scalar_t(0) ? a : a * negval;
    });
  });
}

void leaky_relu_backward_kernel(TensorIteratorBase& iter, const Scalar& negval_) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "leaky_relu_backward_cuda", [&]() {
    auto negval = negval_.to<scalar_t>();
    gpu_kernel(iter, [negval]GPU_LAMBDA(scalar_t a, scalar_t b) -> scalar_t {
      return a > scalar_t(0) ? b : b * negval;
    });
  });
}

void hardswish_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return x * std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardswish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardswish_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_half(0.5f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_half]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        if (self_val < neg_three) {
          return zero;
        } else if (self_val <= three) {
          return grad_val * ((self_val / three) + one_half);
        } else {
          return grad_val;
        }
    });
  });
}

void hardsigmoid_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    const T_ACC three(3.0f);
    const T_ACC six(6.0f);
    gpu_kernel(iter, [zero, one_sixth, three, six]GPU_LAMBDA(scalar_t self_val) -> scalar_t {
      T_ACC x = static_cast<T_ACC>(self_val);
      return std::min(std::max(x + three, zero), six) * one_sixth;
    });
  });
}

void hardsigmoid_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "hardsigmoid_backward_cuda", [&]() {
    using T_ACC = acc_type<scalar_t, true>;
    const T_ACC zero(0.0f);
    const T_ACC three(3.0f);
    const T_ACC neg_three(-3.0f);
    const T_ACC one_sixth(1.0f / 6.0f);
    gpu_kernel(
      iter,
      [zero, three, neg_three, one_sixth]GPU_LAMBDA(scalar_t grad_val_, scalar_t self_val_) -> scalar_t {
        T_ACC grad_val = static_cast<T_ACC>(grad_val_);
        T_ACC self_val = static_cast<T_ACC>(self_val_);
        return (self_val > neg_three && self_val < three)
          ? grad_val * one_sixth
          : zero;
    });
  });
}

void silu_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC x_acc = static_cast<T_ACC>(x);
              return x_acc / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
            });
      });
}

void silu_backward_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "silu_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
              using T_ACC = acc_type<scalar_t, true>;
              const T_ACC dy_acc = static_cast<T_ACC>(dy);
              const T_ACC x_acc = static_cast<T_ACC>(x);
              const T_ACC s_acc =
                  T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
              return dy_acc * s_acc * (T_ACC(1) + x_acc * (T_ACC(1) - s_acc));
            });
      });
}

void mish_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC x_acc = static_cast<T_ACC>(x);
          return x_acc * c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
      });
      });
}

void mish_backward_kernel(TensorIterator& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      iter.dtype(),
      "mish_backward_cuda",
      [&]() {
        gpu_kernel(
            iter,
            [] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
          using T_ACC = acc_type<scalar_t, true>;
          const T_ACC dy_acc = static_cast<T_ACC>(dy);
          const T_ACC x_acc = static_cast<T_ACC>(x);
          const T_ACC s_acc =
              T_ACC(1) / (T_ACC(1) + c10::cuda::compat::exp(-x_acc));
          const T_ACC t_acc =
              c10::cuda::compat::tanh(c10::cuda::compat::log1p(c10::cuda::compat::exp(x_acc)));
          return dy_acc * (t_acc + x_acc * s_acc * (T_ACC(1) - t_acc * t_acc));
      });
      });
}

} // namespace

REGISTER_DISPATCH(hardtanh_backward_stub, &hardtanh_backward_kernel);
REGISTER_DISPATCH(hardshrink_stub, &hardshrink_kernel);
REGISTER_DISPATCH(log_sigmoid_backward_stub, &log_sigmoid_backward_kernel);
REGISTER_DISPATCH(softshrink_stub, &softshrink_kernel);
REGISTER_DISPATCH(shrink_backward_stub, &shrink_backward_kernel);
REGISTER_DISPATCH(elu_stub, &elu_kernel);
REGISTER_DISPATCH(elu_backward_stub, &elu_backward_kernel);
REGISTER_DISPATCH(glu_stub, &glu_kernel);
REGISTER_DISPATCH(leaky_relu_stub, &leaky_relu_kernel);
REGISTER_DISPATCH(leaky_relu_backward_stub, &leaky_relu_backward_kernel);
REGISTER_DISPATCH(hardswish_stub, &hardswish_kernel);
REGISTER_DISPATCH(hardswish_backward_stub, &hardswish_backward_kernel);
REGISTER_DISPATCH(hardsigmoid_stub, &hardsigmoid_kernel);
REGISTER_DISPATCH(hardsigmoid_backward_stub, &hardsigmoid_backward_kernel);
REGISTER_DISPATCH(softplus_stub, &softplus_kernel);
REGISTER_DISPATCH(softplus_backward_stub, &softplus_backward_kernel);
REGISTER_DISPATCH(silu_stub, &silu_kernel);
REGISTER_DISPATCH(silu_backward_stub, &silu_backward_kernel);
REGISTER_DISPATCH(mish_stub, &mish_kernel);
REGISTER_DISPATCH(mish_backward_stub, &mish_backward_kernel);
REGISTER_DISPATCH(threshold_stub, &threshold_kernel_cuda);

} // namespace native
} // namespace at
