#include "hip/hip_runtime.h"
// Adapted from interp.cpp from Caffe util by Pauline Luc
// Originally developed by George Papandreou
#include <ATen/ATen.h>
#include <ATen/ceil_div.h>
#include <ATen/AccumulateType.h>
#include <ATen/NativeFunctions.h>
#include <ATen/TensorUtils.h>
#include <ATen/Utils.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/UpSample.cuh>
#include <ATen/native/cuda/KernelUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>

namespace at {
namespace native {
namespace {

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_out_frame(
    const int n,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const PackedTensorAccessor<scalar_t, 4> idata,
    PackedTensorAccessor<scalar_t, 4> odata) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;

  const int batchsize = idata.size(0);
  const int channels = idata.size(1);
  const int height1 = idata.size(2);
  const int width1 = idata.size(3);
  const int height2 = odata.size(2);
  const int width2 = odata.size(3);

  if (index < n) {
    const int w2 = index % width2; // 0:width2-1
    const int h2 = index / width2; // 0:height2-1

    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    for (int n = 0; n < batchsize; n++) {
      for (int c = 0; c < channels; ++c) {
        const accscalar_t val = h0lambda *
                (w0lambda * idata[n][c][h1][w1] +
                 w1lambda * idata[n][c][h1][w1 + w1p]) +
            h1lambda *
                (w0lambda * idata[n][c][h1 + h1p][w1] +
                 w1lambda * idata[n][c][h1 + h1p][w1 + w1p]);
        odata[n][c][h2][w2] = static_cast<scalar_t>(val);
      }
    }
  }
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_nhwc_out_frame(
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    const int batchsize,
    const int channels,
    const int height1,
    const int width1,
    const int height2,
    const int width2,
    const scalar_t* idata,
    scalar_t* odata,
    const int out_numel) {

  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < out_numel) {
    const int c = index % channels;
    const int w2 = (index / channels) % width2;
    const int h2 = (index / channels / width2) % height2;
    const int n = index / channels / width2 / height2;

    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;

    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;

    const accscalar_t val = h0lambda * (
        w0lambda * idata[idx_cl(n, h1, w1, c, height1, width1, channels)] +
        w1lambda * idata[idx_cl(n, h1, w1 + w1p, c, height1, width1, channels)]
      ) + h1lambda * (
        w0lambda * idata[idx_cl(n, h1 + h1p, w1, c, height1, width1, channels)] +
        w1lambda * idata[idx_cl(n, h1 + h1p, w1 + w1p, c, height1, width1, channels)]
      );
    odata[idx_cl(n, h2, w2, c, height2, width2, channels)] = static_cast<scalar_t>(val);
  }
}

// Backward (adjoint) operation 1 <- 2 (accumulates)
template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_backward_out_frame(
    const size_t nc,
    const int height1,
    const int width1,
    const int height2,
    const int width2,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    scalar_t* __restrict__ idata,
    const scalar_t* __restrict__ odata) {
  const size_t o_numel = nc * width2 * height2;
  const size_t i_numel = nc * width1 * height1;
  for (size_t index = blockDim.x * blockIdx.x + threadIdx.x; index < o_numel;
       index += blockDim.x * gridDim.x) {
    size_t index_temp = index;
    const int w2 = index_temp % width2; // 0:width2-1
    index_temp /= width2;
    const int h2 = index_temp % height2; // 0:height2-1
    const size_t nc = index_temp / height2;
    //
    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;
    //
    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;
    //
    const scalar_t d2val = odata[index];
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1, w1),
        i_numel,
        static_cast<scalar_t>(h0lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1, w1 + w1p),
        i_numel,
        static_cast<scalar_t>(h0lambda * w1lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1 + h1p, w1),
        i_numel,
        static_cast<scalar_t>(h1lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx(nc, height1, width1, h1 + h1p, w1 + w1p),
        i_numel,
        static_cast<scalar_t>(h1lambda * w1lambda * d2val),
        true);
  }
}

template <typename scalar_t, typename accscalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void upsample_bilinear2d_backward_nhwc_out_frame(
    const size_t nc,
    const int height1,
    const int width1,
    const int height2,
    const int width2,
    const accscalar_t rheight,
    const accscalar_t rwidth,
    const bool align_corners,
    scalar_t* __restrict__ idata,
    const scalar_t* __restrict__ odata,
    const int channels,
    const size_t o_numel,
    const size_t i_numel) {

  const int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < o_numel) {
    const int c = index % channels;
    const int w2 = (index / channels) % width2;
    const int h2 = (index / channels / width2) % height2;
    const int n = index / channels / width2 / height2;

    const accscalar_t h1r = area_pixel_compute_source_index<accscalar_t>(
        rheight, h2, align_corners, /*cubic=*/false);
    const int h1 = h1r;
    const int h1p = (h1 < height1 - 1) ? 1 : 0;
    const accscalar_t h1lambda = h1r - h1;
    const accscalar_t h0lambda = static_cast<accscalar_t>(1) - h1lambda;

    const accscalar_t w1r = area_pixel_compute_source_index<accscalar_t>(
        rwidth, w2, align_corners, /*cubic=*/false);
    const int w1 = w1r;
    const int w1p = (w1 < width1 - 1) ? 1 : 0;
    const accscalar_t w1lambda = w1r - w1;
    const accscalar_t w0lambda = static_cast<accscalar_t>(1) - w1lambda;

    const scalar_t d2val = odata[index];
    fastAtomicAdd(
        idata,
        idx_cl(n, h1, w1, c, height1, width1, channels),
        i_numel,
        static_cast<scalar_t>(h0lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx_cl(n, h1, w1 + w1p, c, height1, width1, channels),
        i_numel,
        static_cast<scalar_t>(h0lambda * w1lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx_cl(n, h1 + h1p, w1, c, height1, width1, channels),
        i_numel,
        static_cast<scalar_t>(h1lambda * w0lambda * d2val),
        true);
    fastAtomicAdd(
        idata,
        idx_cl(n, h1 + h1p, w1 + w1p, c, height1, width1, channels),
        i_numel,
        static_cast<scalar_t>(h1lambda * w1lambda * d2val),
        true);
  }
}

static void upsample_bilinear2d_out_cuda_template(
    const Tensor& output,
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg input_arg{input, "input", 1}, output_arg{output, "output", 2};
  checkAllSameGPU(__func__, {input_arg, output_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input.size(0);
  int channels = input.size(1);
  int input_height = input.size(2);
  int input_width = input.size(3);

  const auto memory_format = input.suggest_memory_format();

  if (input.sizes() == output.sizes()) {
    output.copy_(input);
    return;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "upsample_bilinear2d_out_frame", [&] {
    // heuristic: only use channels_last path when it's faster than the contiguous path
    if (memory_format == at::MemoryFormat::ChannelsLast && channels >= 16 && \
          output.is_contiguous(memory_format)) {
      using accscalar_t = at::acc_type<scalar_t, true>;

      TORCH_CHECK(input.numel() < std::numeric_limits<int>::max(),
        "upsample_bilinear2d_nhwc only supports input tensors with less than INT_MAX elements");
      TORCH_CHECK(output.numel() < std::numeric_limits<int>::max(),
        "upsample_bilinear2d_nhwc only supports output tensors with less than INT_MAX elements");

      const int batchsize = input.size(0);
      const int channels = input.size(1);
      const int height1 = input.size(2);
      const int width1 = input.size(3);
      const int height2 = output.size(2);
      const int width2 = output.size(3);

      // const int num_kernels = output_height * output_width;
      const int num_kernels = output.numel();
      const int num_threads = std::min(
          at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);

      at::Tensor input_cl = input.contiguous(at::MemoryFormat::ChannelsLast);

      const scalar_t* idata = input_cl.data_ptr<scalar_t>();
      scalar_t* odata = output.data_ptr<scalar_t>();

      const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
          input_height, output_height, align_corners, scales_h);
      const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales_w);

      upsample_bilinear2d_nhwc_out_frame<scalar_t, accscalar_t>
        <<<ceil_div(num_kernels, num_threads), num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
          rheight, rwidth, align_corners,
          batchsize,
          channels,
          height1,
          width1,
          height2,
          width2,
          idata, odata,
          output.numel());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      // non-channels_last case, not necessarily contiguous
      const int num_kernels = output_height * output_width;
      const int num_threads = std::min(
          at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
      hipStream_t stream = at::cuda::getCurrentCUDAStream();

      using accscalar_t = at::acc_type<scalar_t, true>;

      auto idata = input.packed_accessor64<scalar_t, 4>();
      auto odata = output.packed_accessor64<scalar_t, 4>();

      const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
          input_height, output_height, align_corners, scales_h);
      const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales_w);

      upsample_bilinear2d_out_frame<scalar_t, accscalar_t>
          <<<ceil_div(num_kernels, num_threads),
             num_threads,
             0,
             stream>>>(
              num_kernels, rheight, rwidth, align_corners, idata, odata);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  });
}

static void upsample_bilinear2d_backward_out_cuda_template(
    const Tensor& grad_input,
    const Tensor& grad_output_,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w) {
  TensorArg grad_input_arg{grad_input, "grad_input", 1},
      grad_output_arg{grad_output_, "grad_output_", 2};
  checkAllSameGPU(__func__, {grad_output_arg, grad_input_arg});

  int output_height = output_size[0];
  int output_width = output_size[1];

  int nbatch = input_size[0];
  int channels = input_size[1];
  int input_height = input_size[2];
  int input_width = input_size[3];

  if (grad_input.numel() == 0) {
    return;
  }

  const auto memory_format = grad_output_.suggest_memory_format();

  // initialization to zero is required here. As we launch one thread per output
  // element, and atomicAdd to input gradient. Given a sparse sampling case, our
  // threads are not covering the whole input tensor.
  grad_input.zero_();

  const size_t num_kernels = nbatch * channels * output_height * output_width;
  const int num_threads = std::min(
      at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (grad_output_.sizes() == grad_input.sizes()) {
    grad_input.copy_(grad_output_);
    return;
  }

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad_output_.scalar_type(), "upsample_bilinear2d_backward_out_frame", [&] {
    if (memory_format == at::MemoryFormat::ChannelsLast && channels >= 4 && \
          grad_input.is_contiguous(memory_format)) {
      using accscalar_t = at::acc_type<scalar_t, true>;

      Tensor grad_output = grad_output_.contiguous(at::MemoryFormat::ChannelsLast);

      auto idata = grad_input.data_ptr<scalar_t>();
      auto odata = grad_output.data_ptr<scalar_t>();

      const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
          input_height, output_height, align_corners, scales_h);
      const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales_w);

      upsample_bilinear2d_backward_nhwc_out_frame<scalar_t, accscalar_t>
          <<<ceil_div(num_kernels, static_cast<size_t>(num_threads)), num_threads, 0, stream>>>(
              nbatch * channels,
              input_height,
              input_width,
              output_height,
              output_width,
              rheight,
              rwidth,
              align_corners,
              idata,
              odata,
              channels,
              grad_output.numel(),
              grad_input.numel());
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      using accscalar_t = at::acc_type<scalar_t, true>;

      // This is needed for non-contiguous tensors.
      Tensor grad_input_c = grad_input.is_contiguous() ? grad_input : at::zeros(grad_input.sizes(), grad_input.options());
      Tensor grad_output = grad_output_.contiguous();

      auto idata = grad_input_c.data_ptr<scalar_t>();
      auto odata = grad_output.data_ptr<scalar_t>();

      const accscalar_t rheight = area_pixel_compute_scale<accscalar_t>(
          input_height, output_height, align_corners, scales_h);
      const accscalar_t rwidth = area_pixel_compute_scale<accscalar_t>(
          input_width, output_width, align_corners, scales_w);

      upsample_bilinear2d_backward_out_frame<scalar_t, accscalar_t>
          <<<ceil_div(num_kernels, static_cast<size_t>(num_threads)),
             num_threads,
             0,
             stream>>>(
              nbatch * channels,
              input_height,
              input_width,
              output_height,
              output_width,
              rheight,
              rwidth,
              align_corners,
              idata,
              odata);
      C10_CUDA_KERNEL_LAUNCH_CHECK();

      if (!grad_input.is_contiguous()) {
          grad_input.copy_(grad_input_c);
      }
    }
  });
}

} // namespace

TORCH_IMPL_FUNC(upsample_bilinear2d_out_cuda) (
    const Tensor& input,
    IntArrayRef output_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w,
    const Tensor& output) {
  upsample_bilinear2d_out_cuda_template(output, input, output_size, align_corners, scales_h, scales_w);
}

TORCH_IMPL_FUNC(upsample_bilinear2d_backward_out_cuda) (
    const Tensor& grad_output,
    IntArrayRef output_size,
    IntArrayRef input_size,
    bool align_corners,
    c10::optional<double> scales_h,
    c10::optional<double> scales_w,
    const Tensor& grad_input) {
  // See Note [Writing Nondeterministic Operations]
  // Nondeterministic because of atomicAdd usage
  globalContext().alertNotDeterministic("upsample_bilinear2d_backward_out_cuda");
  upsample_bilinear2d_backward_out_cuda_template(
      grad_input, grad_output, output_size, input_size, align_corners, scales_h, scales_w);
}

} // namespace native
} // namespace at
