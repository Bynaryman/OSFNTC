#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/UnaryOps.h>

#include <limits>

#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/Math.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/NumericUtils.h>
#include <c10/cuda/CUDAMathCompat.h>
#include <c10/core/Scalar.h>
#include <c10/util/complex.h>

namespace at {
namespace native {

void bitwise_not_kernel_cuda(TensorIteratorBase& iter) {
  if (iter.dtype() == ScalarType::Bool) {
    gpu_kernel(iter, []GPU_LAMBDA(bool a) {
      return !a;
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(iter.dtype(), "bitwise_not_cuda", [&]() {
      gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
        return ~a;
      });
    });
  }
}

void exp_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.common_dtype(), "exp_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return std::exp(a);
    });
  });
}

void expm1_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      ScalarType::BFloat16, ScalarType::Half,
      iter.common_dtype(), "expm1_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          return ::expm1(a);
        });
      });
}

// We manually overload rsqrt because std::rsqrt does not work with complex types.
template<typename scalar_t>
__host__ __device__ static inline scalar_t rsqrt_wrapper(scalar_t v) {
  return ::rsqrt(v);
}

template<typename T>
__host__ __device__ static inline c10::complex<T> rsqrt_wrapper(c10::complex<T> v) {
  const c10::complex<T> one = c10::complex<T>(1.0, 0);
  // std::sqrt for c10::complex is overloaded in c10/util/complex_math.h
  return one / ::sqrt(v);
}

void rsqrt_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(
      ScalarType::BFloat16, ScalarType::Half,
      iter.common_dtype(), "rsqrt_cuda",
      [&]() {
        gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
          // In CUDA, ::rsqrt is overloaded for float and at::Half here is implicitly cast to float.
          return rsqrt_wrapper(a);
        });
      });
}

void sqrt_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND2(ScalarType::Half, ScalarType::BFloat16, iter.common_dtype(), "sqrt_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::sqrt(a);
    });
  });
}

void clamp_kernel_cuda(TensorIteratorBase& iter, const Scalar& min_value, const Scalar& max_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(::max(v, lower), upper);
      }
    });
  });
}

void clamp_min_kernel_cuda(TensorIteratorBase& iter, const Scalar& min_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_min_cuda", [&]() {
    auto lower = min_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::max(v, lower);
      }
    });
  });
}

void clamp_max_kernel_cuda(TensorIteratorBase& iter, const Scalar& max_value) {
  AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "clamp_max_cuda", [&]() {
    auto upper = max_value.to<scalar_t>();
    gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t v) -> scalar_t {
      // Propagate nan, which doesn't propagate automatically for ROCm
      if (_isnan(v)) {
        return v;
      } else {
        return ::min(v, upper);
      }
    });
  });
}

void nan_to_num_kernel_cuda(
    TensorIteratorBase& iter,
    c10::optional<double> nan,
    c10::optional<double> pos_inf,
    c10::optional<double> neg_inf) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kHalf, kBFloat16, iter.dtype(), "nan_to_num_cuda", [&]() {
    scalar_t nan_replacement = static_cast<scalar_t>(nan.value_or(0.));
    scalar_t pos_inf_replacement = pos_inf.has_value()
        ? static_cast<scalar_t>(pos_inf.value())
        : std::numeric_limits<scalar_t>::max();
    scalar_t neg_inf_replacement = neg_inf.has_value()
        ? static_cast<scalar_t>(neg_inf.value())
        : std::numeric_limits<scalar_t>::lowest();
    gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
      return (
          at::_isnan(a)
              ? nan_replacement
              : (a == std::numeric_limits<scalar_t>::infinity()
                     ? pos_inf_replacement
                     : (a == -std::numeric_limits<scalar_t>::infinity()
                            ? neg_inf_replacement
                            : a)));
    });
  });
}

void frexp_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND(ScalarType::Half,
    // The iter.dtype() here is the dtype of mantissa output.
    // It's a floating point type and must be the same as the input's dtype.
    iter.dtype(),
    "frexp_cuda", [&]() {
      gpu_kernel_multiple_outputs(iter, [=] GPU_LAMBDA (scalar_t a) -> thrust::tuple<scalar_t, int32_t> {
        int32_t exponent;
        scalar_t mantissa = std::frexp(a, &exponent);
        return {mantissa, exponent};
      });
  });
}

REGISTER_DISPATCH(bitwise_not_stub, &bitwise_not_kernel_cuda);
REGISTER_DISPATCH(exp_stub, &exp_kernel_cuda);
REGISTER_DISPATCH(expm1_stub, &expm1_kernel_cuda);
REGISTER_DISPATCH(rsqrt_stub, &rsqrt_kernel_cuda);
REGISTER_DISPATCH(sqrt_stub, &sqrt_kernel_cuda);
REGISTER_DISPATCH(nan_to_num_stub, &nan_to_num_kernel_cuda);
REGISTER_DISPATCH(frexp_stub, &frexp_kernel_cuda);

} // namespace native
} // namespace at
