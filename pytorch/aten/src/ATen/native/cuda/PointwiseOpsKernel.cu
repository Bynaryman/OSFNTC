#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/AccumulateType.h>
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/PointwiseOps.h>
#include <c10/core/Scalar.h>

namespace at { namespace native {

void addcmul_cuda_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.dtype(), "addcmul_cuda", [&]() {
    // note(mkozuki): If scalar_t is fp16 or bfloat16, cast scalar to float
    // and do math in fp32 for better accuracy.
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto alpha = value.to<accscalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return a + alpha * (static_cast<accscalar_t>(b) * static_cast<accscalar_t>(c));
    });
  });
}

void addcdiv_cuda_kernel(TensorIteratorBase& iter, const Scalar& value) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(kHalf, kBFloat16, iter.dtype(), "addcdiv_cuda", [&]() {
    // note(mkozuki): If scalar_t is fp16 or bfloat16, cast scalar to float
    // and do math in fp32 for better accuracy.
    using accscalar_t = at::acc_type<scalar_t, true>;
    auto alpha = value.to<accscalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return a + alpha * (b / static_cast<accscalar_t>(c));
    });
  });
}

void smooth_l1_backward_cuda_kernel(TensorIterator& iter, const Scalar& norm, double beta) {
  AT_DISPATCH_ALL_TYPES_AND(kHalf, iter.dtype(), "smooth_l1_backward_cuda", [&iter, &norm, beta] {
      auto norm_val = norm.to<scalar_t>();
      scalar_t beta_val(beta);
      gpu_kernel(iter, [norm_val, beta_val]GPU_LAMBDA(scalar_t input, scalar_t target, scalar_t grad_output) -> scalar_t {
        const auto x = input - target;
        if (x < -beta_val)
          return -norm_val * grad_output;
        else if (x > beta_val)
          return norm_val * grad_output;
        else
          return norm_val * x * grad_output / beta_val;
    });
  });
}

void huber_backward_cuda_kernel(TensorIterator& iter, const Scalar& norm, double delta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(kBFloat16, kHalf, iter.dtype(), "huber_backward_cuda", [&iter, &norm, delta] {
    auto norm_val = norm.to<scalar_t>();
    scalar_t delta_val(delta);
    gpu_kernel(iter, [norm_val, delta_val]GPU_LAMBDA(scalar_t input, scalar_t target, scalar_t grad_output) -> scalar_t {
      const auto x = input - target;
      if (x < -delta_val) {
        return -norm_val * grad_output * delta_val;
      } else if (x > delta_val) {
        return norm_val * grad_output * delta_val;
      } else {
        return norm_val * x * grad_output;
      }
    });
  });
}

void mse_backward_cuda_kernel(TensorIterator& iter, const Scalar& value) {
  AT_DISPATCH_FLOATING_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "mse_backward_cuda", [&]() {
    auto alpha = value.to<scalar_t>();
    gpu_kernel(iter, [alpha]GPU_LAMBDA(scalar_t a, scalar_t b, scalar_t c) -> scalar_t {
      return alpha * (a - b) * c;
    });
  });
}

REGISTER_DISPATCH(addcdiv_stub, &addcdiv_cuda_kernel);
REGISTER_DISPATCH(addcmul_stub, &addcmul_cuda_kernel);
REGISTER_DISPATCH(smooth_l1_backward_stub, &smooth_l1_backward_cuda_kernel);
REGISTER_DISPATCH(huber_backward_stub, &huber_backward_cuda_kernel);
REGISTER_DISPATCH(mse_backward_stub, &mse_backward_cuda_kernel);
}} // namespace at::native
