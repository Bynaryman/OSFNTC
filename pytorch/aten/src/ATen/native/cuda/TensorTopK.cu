#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/native/cuda/TensorTopK.h>
#include <ATen/core/TensorBase.h>
#include <ATen/ceil_div.h>
#include <ATen/Dispatch.h>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/cuda/ScanUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/native/cuda/SortingRadixSelect.cuh>
#include <ATen/native/cuda/SortUtils.cuh>

#include <c10/macros/Macros.h>

using namespace at::native;

namespace at {
namespace native {
namespace {

template <typename T>
struct AddOp {
  __device__ __forceinline__ T operator()(T const &lhs, T const &rhs) {
    return (lhs + rhs);
  }
};

template <typename T, typename IndexType, int Dim, bool Order>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void gatherTopK(at::cuda::detail::TensorInfo<T, IndexType> input,
                           IndexType inputSliceSize,
                           IndexType outputSliceSize, // aka `k`

                           IndexType numInputSlices,
                           IndexType inputWithinSliceStride,

                           at::cuda::detail::TensorInfo<T, IndexType> topK,
                           IndexType numTopKSlices,
                           IndexType topKWithinSliceStride,

                           at::cuda::detail::TensorInfo<int64_t, IndexType> indices,
                           IndexType indicesWithinSliceStride) {
  // Indices are limited to integer fp precision, so counts can fit in
  // int32, regardless of IndexType
#if defined(USE_ROCM)
  __shared__ int smem[64];
#else
  __shared__ int smem[32]; // one per each warp, up to warp limit
#endif
  IndexType slice = getLinearBlockId<IndexType>();
  if (slice >= numInputSlices) {
    return;
  }

  // Find the start offset for our slice
  IndexType sliceStartIndex =
    at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, input);
  IndexType topKSliceStartIndex =
    at::cuda::detail::IndexToOffset<T, IndexType, Dim>::get(slice, topK);
  IndexType indicesSliceStartIndex =
    at::cuda::detail::IndexToOffset<int64_t, IndexType, Dim>::get(slice, indices);

  T* inputSliceStart = &input.data[sliceStartIndex];
  T* topKSliceStart = &topK.data[topKSliceStartIndex];
  int64_t* indicesSliceStart = &indices.data[indicesSliceStartIndex];

  // Find the k-th highest element in our input
  T topKValue = static_cast<T>(0);
  radixSelect<T, typename TopKTypeConfig<T>::RadixType, IndexType, Order>(
    inputSliceStart, outputSliceSize,
    inputSliceSize, inputWithinSliceStride,
    smem, &topKValue);
  const auto topKConverted = at::native::TopKTypeConfig<T>::convert(topKValue);

  // Every value that is strictly less/greater than `pattern`
  // (depending on sort dir) in sorted int format is in the top-K.
  // The top-K value itself might not be unique.
  //
  // Since there are a variable number of elements that we see that
  // are within the top-k, we don't know at what index to write out
  // the resulting values.
  // In order to get this, we perform an exclusive prefix sum of
  // `hasTopK`. This will return the resulting index into which we
  // need to write the result, if a thread has a result.

  // All threads need to participate in the loop and the prefix sum,
  // but not necessarily in the load; hence loop bounds being rounded
  // up to a multiple of the block dim.
  IndexType numIterations = round_up(inputSliceSize, (IndexType) blockDim.x);
  IndexType writeIndexStart = 0;

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK;
    if (Order) {
      hasTopK = inRange && (convertedV > topKConverted);
    } else {
      hasTopK = inRange && (convertedV < topKConverted);
    }

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(
        smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    writeIndexStart += carry;
  }

  // We need to fill in the rest with actual == top-K values.
  // The number that we need is outputSliceSize -
  // writeIndexStart. There might be more than that number available,
  // in which case we have to choose the first seen set. We do this
  // via a prefix sum to calculate indices for writing results.
  CUDA_KERNEL_ASSERT(outputSliceSize >= writeIndexStart);
  IndexType topKRemaining = (outputSliceSize - writeIndexStart);

  for (IndexType i = threadIdx.x; i < numIterations; i += blockDim.x) {
    bool inRange = (i < inputSliceSize);
    T v =
      inRange ? doLdg(&inputSliceStart[i * inputWithinSliceStride]) : static_cast<T>(0);
    const auto convertedV = at::native::TopKTypeConfig<T>::convert(v);
    bool hasTopK = inRange && (convertedV == topKConverted);

    int index;
    int carry;
    at::cuda::exclusiveBinaryPrefixScan<int, true>(
        smem, hasTopK, &index, &carry, AddOp<int>());

    if (hasTopK && index < topKRemaining) {
      int writeIndex = writeIndexStart + index;
      CUDA_KERNEL_ASSERT(writeIndex < outputSliceSize);

      IndexType topKOffset = writeIndex * topKWithinSliceStride;
      IndexType indexOffset = writeIndex * indicesWithinSliceStride;

      topKSliceStart[topKOffset] = v;
      indicesSliceStart[indexOffset] = i;
    }

    if (carry >= topKRemaining) {
      break;
    }

    topKRemaining -= carry;
    writeIndexStart += carry;
  }

};

} // namespace

void launch_gather_topk_kernel(
    const TensorBase& self, int64_t k, int64_t dim, bool largest, bool sorted,
    const TensorBase& values, const TensorBase& indices) {
  int numDims = self.dim();
  numDims = numDims == 0 ? 1 : numDims;
  TORCH_CHECK(numDims <= MAX_DIMS, "input tensor has too many dimensions");
  int64_t sliceSize = self.dim() == 0 ? 1 : self.size(dim);

  auto input = self.contiguous();
  // static_cast is required to ensure that the correct type (INDEX_T)
  // is provided to the kernel for the arguments.

#define RUN_K(INDEX_T, DIM, DIR)                                        \
  gatherTopK<scalar_t, INDEX_T, DIM, DIR>                               \
    <<<grid, block, 0, c10::cuda::getCurrentCUDAStream()>>>(            \
      inputInfo,                                                        \
      static_cast<INDEX_T>(sliceSize),                                  \
      static_cast<INDEX_T>(k),                                          \
      static_cast<INDEX_T>(inputSlices),                                \
      /* The actual dimension that the k-selection is running in */     \
      /* may have changed from collapseDims() */                        \
      static_cast<INDEX_T>(inputInfo.strides[collapseInputDim]),        \
      topKInfo,                                                         \
      static_cast<INDEX_T>(topKSlices),                                 \
      static_cast<INDEX_T>(topKInfo.strides[collapseTopKDim]),          \
      indicesInfo,                                                      \
      static_cast<INDEX_T>(indicesInfo.strides[collapseIndicesDim]));   \
  C10_CUDA_KERNEL_LAUNCH_CHECK();

#define RUN_DIR(INDEX_T, DIM)                   \
  if (largest) {                                \
    RUN_K(INDEX_T, DIM, true);                  \
  } else {                                      \
    RUN_K(INDEX_T, DIM, false);                 \
  }

#define RUN_DIM(INDEX_T)                        \
  if (allDims == 1) {                           \
    RUN_DIR(INDEX_T, 1);                        \
  } else if (allDims == 2) {                    \
    RUN_DIR(INDEX_T, 2);                        \
  } else if (allDims == 3) {                    \
    RUN_DIR(INDEX_T, 3);                        \
  } else {                                      \
    RUN_DIR(INDEX_T, -1);                       \
  }

#define RUN_T(INDEX_T)                                                  \
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, input.scalar_type(), "topk_out_cuda", [&] { \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> inputInfo =           \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(input);          \
    at::cuda::detail::TensorInfo<scalar_t, INDEX_T> topKInfo =            \
      at::cuda::detail::getTensorInfo<scalar_t, INDEX_T>(values);         \
    at::cuda::detail::TensorInfo<int64_t, INDEX_T> indicesInfo =          \
      at::cuda::detail::getTensorInfo<int64_t, INDEX_T>(indices);         \
    /* tensorInfoLegacyIfScalar*/                                         \
    if (!input.dim()) {                                                   \
      inputInfo.dims = 1;                                                 \
      inputInfo.sizes[0] = 1;                                             \
      inputInfo.strides[0] = 1;                                           \
      topKInfo.dims = 1;                                                  \
      topKInfo.sizes[0] = 1;                                              \
      topKInfo.strides[0] = 1;                                            \
      indicesInfo.dims = 1;                                               \
      indicesInfo.sizes[0] = 1;                                           \
      indicesInfo.strides[0] = 1;                                         \
    }                                                                     \
    /* We use these structures solely to find the offset to */            \
    /* each slice we are operating on */                                  \
    inputInfo.sizes[dim] = 1;                                             \
    topKInfo.sizes[dim] = 1;                                              \
    indicesInfo.sizes[dim] = 1;                                           \
    /* stash the stride of dim because it can be accidentally collapsed */ \
    auto strideTopK = topKInfo.strides[dim];                              \
    auto strideIndices = indicesInfo.strides[dim];                        \
    /* Collapse all other dims */                                         \
    int collapseInputDim = inputInfo.collapseDims(dim);                   \
    int collapseTopKDim = topKInfo.collapseDims(dim);                     \
    int collapseIndicesDim = indicesInfo.collapseDims(dim);               \
    /* restore stride in case it was collapsed */                         \
    topKInfo.strides[collapseTopKDim] = strideTopK;                       \
    indicesInfo.strides[collapseIndicesDim] = strideIndices;              \
    int64_t inputSlices = 1;                                              \
    for (int i = 0; i < inputInfo.dims; ++i) {                            \
      inputSlices *= inputInfo.sizes[i];                                  \
    }                                                                     \
    int64_t topKSlices = 1;                                               \
    for (int i = 0; i < topKInfo.dims; ++i) {                             \
      topKSlices *= topKInfo.sizes[i];                                    \
    }                                                                     \
                                                                          \
    dim3 grid;                                                            \
    TORCH_INTERNAL_ASSERT(getGridFromTiles(inputSlices, grid), "Too many slices to sort"); \
                                                                          \
    dim3 block(std::min(at::ceil_div(sliceSize, (int64_t) C10_WARP_SIZE)*(int64_t) C10_WARP_SIZE, (int64_t) 1024)); \
                                                                          \
    /* This is used as a template parameter to calculate indices. */      \
    /* We only specialize it if all collapsed dim sizes are the */        \
    /* same; otherwise, we use -1 which is the specialization */          \
    /* parameter for arbitrary dimensions */                              \
    int allDims = inputInfo.dims;                                         \
    if (topKInfo.dims != allDims || indicesInfo.dims != allDims) {        \
      allDims = -1;                                                       \
    }                                                                     \
                                                                          \
    RUN_DIM(INDEX_T);                                                     \
  });

  // the below is safe with 0-dimensional tensors because it is based on
  // TensorInfo which implicitly expands to 1-dimensional.
  if (input.numel() > 0) {
    // Based on required index size, run the algorithm with the
    // appropriate index type
    if (at::cuda::detail::canUse32BitIndexMath(input) &&
        at::cuda::detail::canUse32BitIndexMath(values) &&
        at::cuda::detail::canUse32BitIndexMath(indices)) {
      RUN_T(uint32_t);
    } else {
      RUN_T(uint64_t);
    }
  }
#undef RUN_T
#undef RUN_DIM
#undef RUN_DIR
#undef RUN_K
}

} // at::native
} // at
