#include "hip/hip_runtime.h"
#include <ATen/native/layer_norm.h>

#include <type_traits>

#include <thrust/tuple.h>

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/NativeFunctions.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/native/cuda/block_reduce.cuh>

#include <c10/cuda/CUDAMathCompat.h>

namespace at {
namespace native {

namespace {

constexpr int kCUDANumThreads = 256;
constexpr int kColwiseReduceTileSize = 32;
constexpr int vec_size = 4; //we could make it dependent on dtype, but that would lead to different results between float and low-p types

// aligned vector generates vectorized load/store on CUDA (copy-pasted from MemoryAccess.cuh)
template<typename scalar_t, int vec_size>
struct alignas(sizeof(scalar_t) * vec_size) aligned_vector {
  scalar_t val[vec_size];
};


template <typename T, typename T_ACC>
__global__ void RowwiseMomentsCUDAKernel(
    int64_t N,
    T_ACC eps,
    const T* X,
    T_ACC* mean,
    T_ACC* rstd) {
  using WelfordType = WelfordData<T_ACC, int64_t, T_ACC>;
  using WelfordOp =
      WelfordOps<T_ACC, T_ACC, int64_t, T_ACC, thrust::pair<T_ACC, T_ACC>>;

  __shared__
      typename std::aligned_storage<sizeof(WelfordType), alignof(WelfordType)>::
          type val_shared[C10_WARP_SIZE];
  WelfordType* val_shared_ptr = reinterpret_cast<WelfordType*>(val_shared);

  const int64_t i = blockIdx.x;
  WelfordOp welford_op = {/*correction=*/0, /*take_sqrt=*/false};
  WelfordType val(0, 0, 0, 0);

  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    val = welford_op.reduce(val, static_cast<T_ACC>(X[index]), index);
  }
  val = cuda_utils::BlockReduce(
      val,
      welford_op,
      /*identity_element=*/WelfordType(0, 0, 0, 0),
      val_shared_ptr);

  if (threadIdx.x == 0) {
    T_ACC m1;
    T_ACC m2;
    thrust::tie(m2, m1) = welford_op.project(val);
    mean[i] = m1;
    rstd[i] = c10::cuda::compat::rsqrt(m2 + eps);
  }
}

template <typename T, typename T_ACC>
__global__ void LayerNormForwardCUDAKernel(
    int64_t N,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    const T* gamma,
    const T* beta,
    T* Y) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    const T_ACC beta_v =
        beta == nullptr ? T_ACC(0) : static_cast<T_ACC>(beta[j]);
    Y[index] = (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
            static_cast<T_ACC>(rstd[i]) * gamma_v +
        beta_v;
  }
}

struct WelfordDataLN{
  float mean;
  float sigma2;
  float count;
  C10_HOST_DEVICE WelfordDataLN(): mean(0.f), sigma2(0.f), count(0.f){}
  C10_HOST_DEVICE WelfordDataLN(float mean, float sigma2, float count): mean(mean), sigma2(sigma2), count(count) {}
};

template<typename U> __device__
WelfordDataLN cuWelfordOnlineSum(
  const U val,
  const WelfordDataLN& curr_sum)
{
  U delta = val - curr_sum.mean;
  U new_count = curr_sum.count + 1.f;
  U new_mean = curr_sum.mean + delta * (1.f/new_count); //proper division is slow, this is less accurate but noticeably faster
  return {new_mean, curr_sum.sigma2 + delta * (val - new_mean), new_count};
}

__device__
WelfordDataLN cuWelfordCombine(
  const WelfordDataLN dataB,
  const WelfordDataLN dataA
) {
  using U = decltype(dataB.count);
  U delta = dataB.mean - dataA.mean;
  U count = dataA.count + dataB.count;
  U mean, sigma2;
  if (count > decltype(dataB.count){0}) {
    auto coef = 1.f/count; //NB we don't use --use_fast_math, but this is emulation, 1./count goes to intrinsic, `* coef` is multiplication, instead of slow fp division
    auto nA = dataA.count * coef;
    auto nB = dataB.count * coef;
    mean = nA*dataA.mean + nB*dataB.mean;
    sigma2 = dataA.sigma2 + dataB.sigma2 + delta * delta * dataA.count * nB;
  } else {
    mean = U(0);
    sigma2 = U(0);
  }
  return {mean, sigma2, count};
}

template<typename T>
__device__ WelfordDataLN compute_stats(
  const T*  __restrict__ X,
  const int N,
  float * buf
  ) {
    //X points to the row to read
    using vec_t = aligned_vector<T, vec_size>;
    using acc_t = acc_type<T, true>;
    const vec_t * X_vec = reinterpret_cast<const vec_t*>(X);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const int n_vec_to_read = N/vec_size;
    WelfordDataLN wd(0.f, 0.f, 0.f);
    //no tail, we check that N is multiple of vec_size
    for (int i = thrx; i < n_vec_to_read; i += numx) {
      vec_t data = X_vec[i];
      #pragma unroll
      for (int ii=0; ii < vec_size; ii++){
        wd = cuWelfordOnlineSum(static_cast<acc_t>(data.val[ii]), wd);
      }
    }
    // intra-warp reduction
    for (int offset = (C10_WARP_SIZE >> 1); offset > 0; offset >>= 1) {
        WelfordDataLN wdB{WARP_SHFL_DOWN(wd.mean, offset),
        WARP_SHFL_DOWN(wd.sigma2, offset), WARP_SHFL_DOWN(wd.count, offset)};
        wd = cuWelfordCombine(wd, wdB);
    }
    // threadIdx.x == 0 has correct values for each warp
    // inter-warp reductions
    if (blockDim.y > 1) {
      float * meansigmabuf = buf;
      float * countbuf = buf + blockDim.y;
      for (int offset = blockDim.y/2;  offset > 0;  offset /= 2) {
        // upper half of warps write to shared
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2*offset) {
          const int wrt_y = threadIdx.y - offset;
          meansigmabuf[2*wrt_y] = wd.mean;
          meansigmabuf[2*wrt_y+1] = wd.sigma2;
          countbuf[wrt_y] = wd.count;
        }
        __syncthreads();
        // lower half merges
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          WelfordDataLN wdB{meansigmabuf[2*threadIdx.y],
                          meansigmabuf[2*threadIdx.y+1],
                          countbuf[threadIdx.y]};
          wd = cuWelfordCombine(wd, wdB);
        }
        __syncthreads();
      }
      if (threadIdx.x == 0 && threadIdx.y ==0) {
        meansigmabuf[0] = wd.mean;
        meansigmabuf[1] = wd.sigma2/float(N);
      }
      __syncthreads();
      return WelfordDataLN{meansigmabuf[0], meansigmabuf[1],0.f};

    } else {
      return WelfordDataLN{WARP_SHFL(wd.mean,0), WARP_SHFL(wd.sigma2,0)/float(N), 0.f};
    }
}


template <typename T, typename T_ACC,
typename std::enable_if<!std::is_same<T, double>::value, int>::type = 0>
__device__ __inline__ void vectorized_layer_norm_kernel_impl(
  const int N,
  T_ACC eps,
  const  T* __restrict__ X,
  const  T* gamma,
  const  T* beta,
  T_ACC* mean,
  T_ACC* rstd,
  T* Y){
    extern __shared__ float s_data[]; //if we made smem WelfordDataLN type, there would be bank conflicts,
    //as one thread would have to write 3 consecutive floats
    auto i1 = blockIdx.x;
    const T * block_row = X + i1 * N;
    WelfordDataLN wd = compute_stats(block_row, N, s_data);
    using vec_t = aligned_vector<T, vec_size>;
    const vec_t * X_vec = reinterpret_cast<const vec_t*>(block_row);
    vec_t * Y_vec = reinterpret_cast<vec_t*>(Y + i1 * N);
    const int numx = blockDim.x * blockDim.y;
    const int thrx = threadIdx.x + threadIdx.y * blockDim.x;
    const int n_vec_to_read = N/vec_size;
    T_ACC rstd_val = c10::cuda::compat::rsqrt(wd.sigma2 + eps);
    //no tail, N is guaranteed to be multiple of vec size
    for (int i = thrx; i < n_vec_to_read; i += numx) {
      vec_t data = X_vec[i];
      vec_t out;
      //computation is performed in T_ACC, X is cast to T_ACC and result is implicitly cast to T
      if (gamma != nullptr && beta != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = static_cast<T_ACC>(gamma[i*vec_size + ii]) * (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean))
          + static_cast<T_ACC>(beta[i*vec_size + ii]);
        }
      } else if (gamma != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = static_cast<T_ACC>(gamma[i*vec_size + ii]) * (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean));
        }
      } else if (beta != nullptr) {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = (rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean)) + static_cast<T_ACC>(beta[i*vec_size + ii]);
        }
      } else {
        #pragma unroll
        for (int ii=0; ii < vec_size; ii++){
          out.val[ii] = rstd_val * (static_cast<T_ACC>(data.val[ii]) - wd.mean);
        }
      }
      Y_vec[i] = out;
    }
    if (thrx == 0) {
      mean[i1] = wd.mean;
      rstd[i1] = rstd_val;
    }
}

template <typename T, typename T_ACC,
typename std::enable_if<std::is_same<T, double>::value, int>::type = 0>
__device__ __inline__ void vectorized_layer_norm_kernel_impl(
  const int N,
  T_ACC eps,
  const  T* __restrict__ X,
  const  T* gamma,
  const  T* beta,
  T_ACC* mean,
  T_ACC* rstd,
  T* Y){
    CUDA_KERNEL_ASSERT("doesn't work with double");
  }

//to avoid windows SFINAE errors
template <typename T, typename T_ACC>
__global__ __inline__ void vectorized_layer_norm_kernel(
  const int N,
  T_ACC eps,
  const  T* __restrict__ X,
  const  T* gamma,
  const  T* beta,
  T_ACC* mean,
  T_ACC* rstd,
  T* Y){
    vectorized_layer_norm_kernel_impl(N, eps, X, gamma, beta, mean, rstd, Y);
  }

template <typename T>
__global__ void ComputeInternalGradientsCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    acc_type<T, true>* ds,
    acc_type<T, true>* db) {
  using T_ACC = acc_type<T, true>;
  __shared__ T_ACC ds_shared[C10_WARP_SIZE];
  __shared__ T_ACC db_shared[C10_WARP_SIZE];
  const int64_t i = blockIdx.x;
  T_ACC sum1 = 0;
  T_ACC sum2 = 0;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    sum1 +=
        static_cast<T_ACC>(dY[index]) * static_cast<T_ACC>(X[index]) * gamma_v;
    sum2 += static_cast<T_ACC>(dY[index]) * gamma_v;
  }
  sum1 = cuda_utils::BlockReduceSum<T_ACC>(sum1, ds_shared);
  sum2 = cuda_utils::BlockReduceSum<T_ACC>(sum2, db_shared);
  if (threadIdx.x == 0) {
    ds[i] = sum1;
    db[i] = sum2;
  }
}


template<typename T, typename T_ACC>
__device__ __inline__ void compute_gI(
  const T* __restrict__ dY,
  const T* __restrict__ X,
  const T_ACC* __restrict__ mean,
  const T_ACC* __restrict__ rstd,
  const T* __restrict__ gamma,
  T* dX,
  const int N,
  T_ACC * buf){
    const auto i1 = blockIdx.x;
    const T_ACC mean_val = mean[i1];
    const T_ACC rstd_val = rstd[i1];
    T_ACC stats_x1{0}, stats_x2{0};
    constexpr int unroll = 4;
    auto l = unroll * threadIdx.x;
    const T * X_i = X + i1 * N;
    const T * dY_i = dY + i1 * N;
    T * dX_i = dX + i1 * N;
    //vectorized reads don't improve perf, so use regular unrolling

    for (; l+unroll - 1 < N; l += blockDim.x * unroll){
      #pragma unroll
      for (int k=0; k< unroll; k++){
          T_ACC gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l+k]) : T_ACC(1);
          const T_ACC c_h = static_cast<T_ACC>(X_i[l+k]);
          const T_ACC c_loss = static_cast<T_ACC>(dY_i[l+k]);
          stats_x1 += c_loss * gamma_val;
          stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
      }
    }
    for (;  l < N; l ++) {
          T_ACC gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);
          const T_ACC c_h = static_cast<T_ACC>(X_i[l]);
          const T_ACC c_loss = static_cast<T_ACC>(dY_i[l]);
          stats_x1 += c_loss * gamma_val;
          stats_x2 += c_loss * gamma_val * (c_h - mean_val) * rstd_val;
    }

    stats_x1 = cuda_utils::BlockReduceSum(stats_x1, buf);
    stats_x2 = cuda_utils::BlockReduceSum(stats_x2, buf);
    if (threadIdx.x == 0) {
      buf[0] = stats_x1;
      buf[1] = stats_x2;
    }
    __syncthreads();
    stats_x1 = buf[0];
    stats_x2 = buf[1];
    T_ACC fH = N;
    T_ACC term1 = (T_ACC(1) / fH) * rstd_val;

    for (int l = threadIdx.x; l < N; l += blockDim.x){
        const T_ACC x = X_i[l];
        const T_ACC dy = dY_i[l];
        T_ACC gamma_val = (gamma != nullptr) ? static_cast<T_ACC>(gamma[l]) : T_ACC(1);
        T_ACC f_grad_input = fH * gamma_val * dy;
        f_grad_input -= (x - mean_val) * rstd_val * stats_x2;
        f_grad_input -= stats_x1;
        f_grad_input *= term1;
        dX_i[l] = f_grad_input;
    }
  }



template<typename T, typename T_ACC>
__global__ void layer_norm_grad_input_kernel(
  const T* __restrict__ dY,
  const T* __restrict__ X,
  const T_ACC* __restrict__ mean,
  const T_ACC* __restrict__ rstd,
  const T* __restrict__ gamma,
  T*  dX,
  const int N){
    alignas(sizeof(double)) extern __shared__ char s_data1[];
    T_ACC * buf = reinterpret_cast<T_ACC*>(&s_data1);

    compute_gI(dY, X, mean, rstd, gamma, dX, N, buf);
  }


template <typename T, typename T_ACC>
__global__ void ComputeGradientFusedParamsCUDAKernel(
    int64_t M,
    int64_t N,
    const T_ACC* mean,
    const T_ACC* rstd,
    const acc_type<T, true>* ds,
    const acc_type<T, true>* db,
    acc_type<T, true>* c1,
    acc_type<T, true>* c2) {
  const int64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < M) {
    const T_ACC s = T_ACC(1) / static_cast<T_ACC>(N);
    const T_ACC a = (db[index] * static_cast<T_ACC>(mean[index]) - ds[index]) *
        static_cast<T_ACC>(rstd[index]) * static_cast<T_ACC>(rstd[index]) *
        static_cast<T_ACC>(rstd[index]) * s;
    c1[index] = a;
    c2[index] =
        -(a * static_cast<T_ACC>(mean[index]) +
          db[index] * static_cast<T_ACC>(rstd[index]) * s);
  }
}

template <typename T, typename T_ACC>
__global__ void LayerNormBackwardCUDAKernel(
    int64_t N,
    const T* dY,
    const T* X,
    const T* gamma,
    const T_ACC* a,
    const acc_type<T, true>* b,
    const acc_type<T, true>* c,
    T* dX) {
  const int64_t i = blockIdx.x;
  for (int64_t j = threadIdx.x; j < N; j += blockDim.x) {
    const int64_t index = i * N + j;
    const T_ACC gamma_v =
        gamma == nullptr ? T_ACC(1) : static_cast<T_ACC>(gamma[j]);
    dX[index] =
        static_cast<T_ACC>(a[i]) * static_cast<T_ACC>(dY[index]) * gamma_v +
        b[i] * static_cast<T_ACC>(X[index]) + c[i];
  }
}

template <typename T, typename T_ACC>
__global__ void GammaBetaBackwardSimpleCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    T* dg,
    T* db) {
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  if (j < N) {
    T_ACC sum1 = 0;
    T_ACC sum2 = 0;
    for (int64_t i = 0; i < M; ++i) {
      const int64_t index = i * N + j;
      sum1 += dg == nullptr ? T_ACC(0)
                            : static_cast<T_ACC>(dY[index]) *
              (static_cast<T_ACC>(X[index]) - static_cast<T_ACC>(mean[i])) *
              static_cast<T_ACC>(rstd[i]);
      sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index]);
    }
    if (dg != nullptr) {
      dg[j] = sum1;
    }
    if (db != nullptr) {
      db[j] = sum2;
    }
  }
}

template <typename T, typename T_ACC>
__global__ void GammaBetaBackwardCUDAKernel1(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    T* dg,
    T* db) {
  __shared__ T_ACC g_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  __shared__ T_ACC b_shared[kColwiseReduceTileSize][kColwiseReduceTileSize + 1];
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  if (j < N) {
    for (int64_t i = threadIdx.y; i < M; i += blockDim.y * 2) {
      const int64_t i1 = i;
      const int64_t i2 = i + blockDim.y;
      const int64_t index1 = i1 * N + j;
      const int64_t index2 = i2 * N + j;
      dg_sum1 += dg == nullptr ? T_ACC(0)
                               : static_cast<T_ACC>(dY[index1]) *
              (static_cast<T_ACC>(X[index1]) - static_cast<T_ACC>(mean[i1])) *
              static_cast<T_ACC>(rstd[i1]);
      db_sum1 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index1]);
      if (i2 < M) {
        dg_sum2 += dg == nullptr ? T_ACC(0)
                                 : static_cast<T_ACC>(dY[index2]) *
                (static_cast<T_ACC>(X[index2]) - static_cast<T_ACC>(mean[i2])) *
                static_cast<T_ACC>(rstd[i2]);
        db_sum2 += db == nullptr ? T_ACC(0) : static_cast<T_ACC>(dY[index2]);
      }
    }
  }
  g_shared[threadIdx.y][threadIdx.x] = dg_sum1;
  g_shared[threadIdx.y + blockDim.y][threadIdx.x] = dg_sum2;
  b_shared[threadIdx.y][threadIdx.x] = db_sum1;
  b_shared[threadIdx.y + blockDim.y][threadIdx.x] = db_sum2;
  __syncthreads();
  T_ACC sum1 = g_shared[threadIdx.x][threadIdx.y];
  T_ACC sum2 = b_shared[threadIdx.x][threadIdx.y];
  sum1 = cuda_utils::WarpReduceSum(sum1);
  sum2 = cuda_utils::WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
  sum1 = g_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum2 = b_shared[threadIdx.x][threadIdx.y + blockDim.y];
  sum1 = cuda_utils::WarpReduceSum(sum1);
  sum2 = cuda_utils::WarpReduceSum(sum2);
  if (threadIdx.x == 0) {
    const int64_t j = blockIdx.x * blockDim.x + threadIdx.y + blockDim.y;
    if (j < N) {
      if (dg != nullptr) {
        dg[j] = sum1;
      }
      if (db != nullptr) {
        db[j] = sum2;
      }
    }
  }
}




template <typename T, typename T_ACC>
__global__ void GammaBetaBackwardCUDAKernel(
    int64_t M,
    int64_t N,
    const T* dY,
    const T* X,
    const T_ACC* mean,
    const T_ACC* rstd,
    T* dg,
    T* db) {
  alignas(sizeof(double)) extern __shared__ char s_data1[];
  T_ACC * s_data_typed = reinterpret_cast<T_ACC*>(&s_data1);
  const int64_t j = blockIdx.x * blockDim.x + threadIdx.x;
  T_ACC dg_sum1 = 0;
  T_ACC dg_sum2 = 0;
  T_ACC db_sum1 = 0;
  T_ACC db_sum2 = 0;
  constexpr int unroll = 8;
  T dYs[unroll];
  T Xs[unroll];
  T_ACC *  means = s_data_typed;
  T_ACC * rstds = s_data_typed + unroll * blockDim.y;
  T_ACC dg_sum = 0;
  T_ACC db_sum = 0;
  if (j < N) {
    int bcounter;
    for (bcounter = 0; bcounter < M/(blockDim.y * unroll); bcounter++){
      int offset = (bcounter * blockDim.y + threadIdx.y) * unroll;
      #pragma unroll
      for (int ii=0; ii<unroll; ii++){
        if (threadIdx.x == 0) {
          means[ii*blockDim.y + threadIdx.y] = mean[offset + ii];
          rstds[ii*blockDim.y + threadIdx.y] = rstd[offset + ii];
        }
        dYs[ii] = dY[(offset + ii) * N + j ];
        Xs[ii] = X[(offset + ii) * N + j];

      }
      __syncthreads();
      #pragma unroll
      for (int ii=0; ii<unroll; ii++){
        dg_sum += dYs[ii] * (Xs[ii] - means[ii*blockDim.y + threadIdx.y]) * rstds[ii * blockDim.y + threadIdx.y];
        db_sum += dYs[ii];
      }
      __syncthreads();
    }
    int offset = (bcounter * blockDim.y + threadIdx.y) * unroll;
    for (int ii = 0; ii<8; ii++ ){
      T_ACC mean_val, rstd_val; // we don't use smem in the tail to avoid awkward synchronizations, perf penalty is negligible
      if ((offset + ii) < M) {
        mean_val = mean[offset+ii];
        rstd_val = rstd[offset+ii];
        dYs[0] = dY[(offset + ii) * N + j ];
        Xs[0] = X[(offset + ii) * N + j];
        dg_sum += dYs[0] * (Xs[0] - mean_val) * rstd_val;
        db_sum += dYs[0];
      }
    }
    s_data_typed[threadIdx.y * blockDim.x + threadIdx.x] = dg_sum;
    s_data_typed[blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] = db_sum;
    __syncthreads();
    for (int offset = blockDim.y/2; offset >=1; offset /= 2){
      if (threadIdx.y < offset) {
        s_data_typed[threadIdx.y * blockDim.x + threadIdx.x] += s_data_typed[(threadIdx.y + offset) * blockDim.x + threadIdx.x];
        s_data_typed[blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x] +=
        s_data_typed[blockDim.x * blockDim.y + (threadIdx.y + offset) * blockDim.x + threadIdx.x];
      }
      __syncthreads();
    }
    if (threadIdx.y == 0) {
      if (dg) {
        dg[j] = s_data_typed[threadIdx.x];
      }
      if (db) {
        db[j] = s_data_typed[threadIdx.x + blockDim.x * blockDim.y];
      }
    }
  }
}

template <typename T, typename T_ACC>
void launch_vectorized_layer_norm_kernel(
  int N,
  int64_t M,
  T_ACC eps,
  const T* X_data,
  const T* gamma_data,
  const T* beta_data,
  T* Y_data,
  T_ACC* mean_data,
  T_ACC* rstd_data
) {
    //constexpr int alignment = 16; //currently unused to make sure float and half results are bw accurate
    auto stream = at::cuda::getCurrentCUDAStream().stream();
    const int num_threads = 128;
    const dim3 threads(C10_WARP_SIZE,num_threads/C10_WARP_SIZE,1);
    const dim3 blocks(M);
    TORCH_INTERNAL_ASSERT_DEBUG_ONLY(threads.y % 2 == 0 || threads.y == 1);
    int nshared = threads.y > 1 ? threads.y * 3/2 *sizeof(T_ACC) : 0;
    vectorized_layer_norm_kernel<<<blocks, threads, nshared, stream>>>(N, eps, X_data,
    gamma_data, beta_data, mean_data, rstd_data, Y_data);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename T, typename T_ACC>
void LayerNormKernelImplInternal(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    T_ACC eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  // assumes input, gamma and beta are of proper shape, this was checked in _check_layer_norm_inputs
  // assumes all tensors are contiguous
  TORCH_CHECK(M <= at::cuda::getCurrentDeviceProperties()->maxGridSize[0], "M should be less than maximum CUDA grid size, \
  file a support request to support bigger batches");
  const T* X_data = X.data_ptr<T>();
  const T* gamma_data = gamma.defined() ? gamma.data_ptr<T>() : nullptr;
  const T* beta_data = beta.defined() ? beta.data_ptr<T>() : nullptr;
  T* Y_data = Y->data_ptr<T>();
  T_ACC* mean_data = mean->data_ptr<T_ACC>();
  T_ACC* rstd_data = rstd->data_ptr<T_ACC>();
  // check if can take fast path - all tensors are properly aligned, N is less than 2^24 (to use float count),
  // N is multiple of vec_size (so that all rows are aligned if tensor is aligned)
  auto can_vectorize = [&](const T * ptr, int alignment){uint64_t addr = reinterpret_cast<uint64_t>(ptr); return addr % alignment == 0;};
  constexpr int num_vec_elems = vec_size;
  constexpr int alignment = num_vec_elems * sizeof(T);
  if ((std::is_same<T, float>::value || std::is_same<T, at::Half>::value) &&
  N <= 1ULL << std::numeric_limits<float>::digits && N % num_vec_elems == 0 &&
  can_vectorize(X_data, alignment) && can_vectorize(Y_data, alignment)) {
    launch_vectorized_layer_norm_kernel(static_cast<int>(N), M, eps, X_data, gamma_data, beta_data, Y_data, mean_data, rstd_data);
  } else {
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  RowwiseMomentsCUDAKernel<T, T_ACC>
      <<<M, cuda_utils::kCUDABlockReduceNumThreads, 0, hip_stream>>>(
          N, eps, X_data, mean_data, rstd_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  LayerNormForwardCUDAKernel<T, T_ACC><<<M, kCUDANumThreads, 0, hip_stream>>>(
      N, X_data, mean_data, rstd_data, gamma_data, beta_data, Y_data);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
  }
}

void LayerNormKernelImpl(
    const Tensor& X,
    const Tensor& gamma,
    const Tensor& beta,
    int64_t M,
    int64_t N,
    double eps,
    Tensor* Y,
    Tensor* mean,
    Tensor* rstd) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "LayerNormKernelImpl",
      [&]() {
        using acc_t = acc_type<scalar_t, true>;
        LayerNormKernelImplInternal<scalar_t, acc_t>(
            X, gamma, beta, M, N, static_cast<acc_t>(eps), Y, mean, rstd);
      });
}

template <typename T>
void LayerNormBackwardKernelImplInternal(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  using T_ACC = acc_type<T, true>;
  TORCH_CHECK(dY.numel() == M * N);
  TORCH_CHECK(mean.numel() == M);
  TORCH_CHECK(rstd.numel() == M);
  TORCH_CHECK(M <= at::cuda::getCurrentDeviceProperties()->maxGridSize[0], "M should be less than maximum CUDA grid size, \
  file a support request to support bigger batches");
  TORCH_CHECK(N <= std::numeric_limits<int>::max(), "Normalized shape should have less than INT_MAX elements, \
  file a support request to support bigger normalized shapes");
  const T* dY_data = dY.template data_ptr<T>();
  const T* X_data = X.template data_ptr<T>();
  const T_ACC* mean_data = mean.template data_ptr<T_ACC>();
  const T_ACC* rstd_data = rstd.template data_ptr<T_ACC>();
  const T* gamma_data =
      gamma.defined() ? gamma.template data_ptr<T>() : nullptr;
  T* dX_data = dX->defined() ? dX->template data_ptr<T>() : nullptr;
  hipStream_t hip_stream = at::cuda::getCurrentCUDAStream();
  if (dX_data != nullptr) {
    const auto kAccType =
        (X.scalar_type() == kHalf || X.scalar_type() == kBFloat16)
        ? kFloat
        : X.scalar_type();
    Tensor ds = at::empty({M}, X.options().dtype(kAccType));
    Tensor db = at::empty({M}, X.options().dtype(kAccType));
    Tensor scale = at::empty({M}, X.options().dtype(kAccType));
    Tensor bias = at::empty({M}, X.options().dtype(kAccType));
    T_ACC* ds_data = ds.template data_ptr<T_ACC>();
    T_ACC* db_data = db.template data_ptr<T_ACC>();
    T_ACC* scale_data = scale.template data_ptr<T_ACC>();
    T_ACC* bias_data = bias.template data_ptr<T_ACC>();
    const int num_threads = 128;
    const dim3 blocks(M);
    int nshared = (num_threads/C10_WARP_SIZE) * sizeof(T_ACC);
    layer_norm_grad_input_kernel<<<blocks, num_threads, nshared, hip_stream>>>(dY_data,
    X_data, mean_data, rstd_data, gamma_data, dX_data, N);
    C10_CUDA_KERNEL_LAUNCH_CHECK();
  }

  if (dgamma->defined() || dbeta->defined()) {
    T* dgamma_data =
        dgamma->defined() ? dgamma->template data_ptr<T>() : nullptr;
    T* dbeta_data = dbeta->defined() ? dbeta->template data_ptr<T>() : nullptr;
    if (M < 512) {
      // For small batch size, do colwise reduce directly.
      const int64_t B = (N + kCUDANumThreads - 1) / kCUDANumThreads;
      GammaBetaBackwardSimpleCUDAKernel<T, T_ACC>
          <<<B, kCUDANumThreads, 0, hip_stream>>>(
              M,
              N,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    } else {
      dim3 threads{16, 32};
      int blocks = (N + threads.x-1)/threads.x;
      GammaBetaBackwardCUDAKernel<T, T_ACC>
          <<<blocks, threads, 2 * sizeof(T_ACC) * threads.x * threads.y, hip_stream>>>(
              M,
              N,
              dY_data,
              X_data,
              mean_data,
              rstd_data,
              dgamma_data,
              dbeta_data);
      C10_CUDA_KERNEL_LAUNCH_CHECK();
    }
  }
}

void LayerNormBackwardKernelImpl(
    const Tensor& dY,
    const Tensor& X,
    const Tensor& mean,
    const Tensor& rstd,
    const Tensor& gamma,
    int64_t M,
    int64_t N,
    Tensor* dX,
    Tensor* dgamma,
    Tensor* dbeta) {
  AT_DISPATCH_FLOATING_TYPES_AND2(
      at::ScalarType::Half,
      at::ScalarType::BFloat16,
      X.scalar_type(),
      "LayerNormBackwardKernelImpl",
      [&]() {
        LayerNormBackwardKernelImplInternal<scalar_t>(
            dY.contiguous(), X, mean, rstd, gamma, M, N, dX, dgamma, dbeta);
      });
}

} // namespace

std::tuple<Tensor, Tensor, Tensor> layer_norm_cuda(
    const Tensor& input,
    IntArrayRef normalized_shape,
    const c10::optional<Tensor>& weight_opt /* optional */,
    const c10::optional<Tensor>& bias_opt /* optional */,
    double eps) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;
  c10::MaybeOwned<Tensor> bias_maybe_owned =
      at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  auto M_N = _check_layer_norm_inputs(input, normalized_shape, weight, bias);
  auto M = M_N.first;
  auto N = M_N.second;
  auto X = input.expect_contiguous();
  auto gamma = weight.expect_contiguous();
  auto beta = bias.expect_contiguous();

  Tensor Y = at::native::empty_like(
      *X,
      c10::nullopt /* dtype */,
      c10::nullopt /* layout */,
      c10::nullopt /* device */,
      c10::nullopt /* pin_memory */,
      LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  auto acc_type = at::toAccumulateType(input.scalar_type(), /*is_cuda=*/true);
  Tensor mean = at::empty({M}, X->options().dtype(acc_type));
  Tensor rstd = at::empty({M}, X->options().dtype(acc_type));
  if (M > 0) {
    LayerNormKernelImpl(*X, *gamma, *beta, M, N, eps, &Y, &mean, &rstd);

    const auto input_shape = input.sizes();
    const size_t axis = input.dim() - normalized_shape.size();

    std::vector<int64_t> stat_shape;
    for (size_t idx = 0; idx < axis; ++idx) {
      stat_shape.push_back(input_shape[idx]);
    }
    for (size_t idx = axis; idx < input.dim(); ++idx) {
      stat_shape.push_back(1);
    }

    mean = mean.view(stat_shape);
    rstd = rstd.view(stat_shape);
  }
  return std::make_tuple(std::move(Y), std::move(mean), std::move(rstd));
}

std::tuple<Tensor, Tensor, Tensor> layer_norm_backward_cuda(
    const Tensor& dY,
    const Tensor& input,
    IntArrayRef normalized_shape,
    const Tensor& mean,
    const Tensor& rstd,
    const c10::optional<Tensor>& weight_opt /* optional */,
    const c10::optional<Tensor>& bias_opt /* optional */,
    std::array<bool, 3> grad_input_mask) {
  // See [Note: hacky wrapper removal for optional tensor]
  c10::MaybeOwned<Tensor> weight_maybe_owned =
      at::borrow_from_optional_tensor(weight_opt);
  const Tensor& weight = *weight_maybe_owned;
  c10::MaybeOwned<Tensor> bias_maybe_owned =
      at::borrow_from_optional_tensor(bias_opt);
  const Tensor& bias = *bias_maybe_owned;

  auto M_N = _check_layer_norm_inputs(input, normalized_shape, weight, bias);
  auto M = M_N.first;
  auto N = M_N.second;
  auto X = input.expect_contiguous();
  auto gamma = weight.expect_contiguous();
  auto beta = bias.expect_contiguous();

  Tensor dX;
  Tensor dgamma;
  Tensor dbeta;
  if (grad_input_mask[0]) {
    dX = at::native::empty_like(
        *X,
        c10::nullopt /* dtype */,
        c10::nullopt /* layout */,
        c10::nullopt /* device */,
        c10::nullopt /* pin_memory */,
        LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[1]) {
    dgamma = M > 0 ? at::native::empty_like(
                         *gamma,
                         c10::nullopt /* dtype */,
                         c10::nullopt /* layout */,
                         c10::nullopt /* device */,
                         c10::nullopt /* pin_memory */,
                         LEGACY_CONTIGUOUS_MEMORY_FORMAT)
                   : at::native::zeros_like(
                         *gamma,
                         c10::nullopt /* dtype */,
                         c10::nullopt /* layout */,
                         c10::nullopt /* device */,
                         c10::nullopt /* pin_memory */,
                         LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (grad_input_mask[2]) {
    dbeta = M > 0 ? at::native::empty_like(
                        *beta,
                        c10::nullopt /* dtype */,
                        c10::nullopt /* layout */,
                        c10::nullopt /* device */,
                        c10::nullopt /* pin_memory */,
                        LEGACY_CONTIGUOUS_MEMORY_FORMAT)
                  : at::native::zeros_like(
                        *beta,
                        c10::nullopt /* dtype */,
                        c10::nullopt /* layout */,
                        c10::nullopt /* device */,
                        c10::nullopt /* pin_memory */,
                        LEGACY_CONTIGUOUS_MEMORY_FORMAT);
  }
  if (M > 0) {
    LayerNormBackwardKernelImpl(
        dY, *X, mean, rstd, *gamma, M, N, &dX, &dgamma, &dbeta);
  }
  return std::make_tuple(std::move(dX), std::move(dgamma), std::move(dbeta));
}


} // namespace native
} // namespace at
