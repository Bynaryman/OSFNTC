#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <limits>
#include <ATen/native/UnaryOps.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/cuda/Math.cuh>
#include <ATen/native/Math.h>

namespace at { namespace native {

void digamma_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "digamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_digamma(a);
    });
  });
}

void trigamma_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "trigamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return calc_trigamma(a);
    });
  });
}

void polygamma_kernel_cuda(TensorIteratorBase& iter, int64_t n) {
  if (n == 0) {
    digamma_kernel_cuda(iter);
  } else if (n == 1) {
    trigamma_kernel_cuda(iter);
  } else {
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "polygamma_cuda", [&]() {
      gpu_kernel(iter, [=] GPU_LAMBDA(scalar_t a) -> scalar_t {
        return calc_polygamma<scalar_t, /*is_cuda=*/true>(int(n), a);
      });
    });
  }
}

void lgamma_kernel_cuda(TensorIteratorBase& iter) {
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(iter.common_dtype(), "lgamma_cuda", [&]() {
    gpu_kernel(iter, []GPU_LAMBDA(scalar_t a) -> scalar_t {
      return ::lgamma(a);
    });
  });
}

REGISTER_DISPATCH(digamma_stub, &digamma_kernel_cuda);
REGISTER_DISPATCH(polygamma_stub, &polygamma_kernel_cuda);
REGISTER_DISPATCH(lgamma_stub, &lgamma_kernel_cuda);

}} // namespace at::native
