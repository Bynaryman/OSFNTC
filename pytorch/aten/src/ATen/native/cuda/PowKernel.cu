#include "hip/hip_runtime.h"
#define TORCH_ASSERT_NO_OPERATORS
#include <ATen/Context.h>
#include <ATen/Dispatch.h>
#include <ATen/native/cuda/Loops.cuh>
#include <ATen/native/DispatchStub.h>
#include <ATen/native/TensorIterator.h>
#include <ATen/native/Pow.h>
#include <c10/core/Scalar.h>

namespace at { namespace native {

// Forward declare some unary kernels
void rsqrt_kernel_cuda(TensorIteratorBase& iter);
void sqrt_kernel_cuda(TensorIteratorBase& iter);
void reciprocal_kernel_cuda(TensorIteratorBase& iter);

namespace {


// SFINAE doesn't work well with NVCC under Windows for math functions like pow and sqrt.
// So we need to define the functions with the explicit function signatures.
// As for pow, the following signatures are defined as the device function:
//   pow(float, int)
//   pow(double, int)
//   pow(float, float)
//   pow(double, double)
#ifdef _MSC_VER
// Functions for pow
// pow for at::Half
static inline __host__ __device__ at::Half pow_(at::Half base, at::Half exp) {
  return static_cast<at::Half>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow for at::BFloat16
static inline __host__ __device__ at::BFloat16 pow_(at::BFloat16 base, at::BFloat16 exp) {
  return static_cast<at::BFloat16>(std::pow(static_cast<float>(base), static_cast<float>(exp)));
}
// pow (floating, floating/int)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<std::is_floating_point<Base_type>::value && (std::is_same<Base_type, Exp_type>::value || std::is_same<Exp_type, int>::value), Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return std::pow(base, exp);
}
// pow (Otherwise)
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ typename std::enable_if<!std::is_same<Base_type, Exp_type>::value && !std::is_same<Exp_type, int>::value, Base_type>::type
  pow_(Base_type base, Exp_type exp) {
  return static_cast<Base_type>(std::pow(static_cast<double>(base), static_cast<double>(exp)));
}
#else
template <typename Base_type, typename Exp_type>
static inline __host__ __device__ Base_type pow_(Base_type base, Exp_type exp) {
  return ::pow(base, exp);
}
#endif

template <typename T>
static inline __host__ __device__ std::enable_if_t<std::is_integral<T>::value, T> pow_(
    T base, T exp) {
  return at::native::powi(base, exp);
}

template <typename T>
static inline __host__ __device__ c10::complex<T> pow_(c10::complex<T> base, c10::complex<T> exp) {
  return c10_complex_math::pow(base, exp);
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar);

template <typename scalar_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, scalar_t base) {
  gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t exp) -> scalar_t {
    return pow_(base, exp);
  });
}

template <typename value_t>
void pow_scalar_tensor_impl(TensorIteratorBase& iter, c10::complex<value_t> base) {
  // For complex, thrust::pow uses the identity
  // pow(a, b) = exp(log(a) * b)
  const auto fct = std::log(base);
  gpu_kernel(iter, [=]GPU_LAMBDA(c10::complex<value_t> exp) -> c10::complex<value_t> {
    return std::exp(fct * exp);
  });
}

void pow_tensor_tensor_kernel(TensorIteratorBase& iter) {
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&] {
    if (iter.is_cpu_scalar(1)) {
      const auto base = iter.scalar_value<scalar_t>(1);
      iter.remove_operand(1);
      pow_scalar_tensor_impl(iter, base);
    } else if (iter.is_cpu_scalar(2)) {
      const auto exp = iter.scalar_value<scalar_t>(2);
      iter.remove_operand(2);
      pow_tensor_scalar_kernel(iter, exp);
    } else {
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base, scalar_t exp) -> scalar_t {
        return pow_(base, exp);
      });
    }
  });
}


template<typename Base_type, typename Exp_type>
void pow_tensor_scalar_kernel_impl(TensorIteratorBase& iter,
                                                 Exp_type exp) {
  const auto d_exp = static_cast<double>(exp);
  // .5 (sqrt), -.5 (rsqrt) and -1 (reciprocal) specializations are handled
  // in pow_tensor_scalar_kernel
  if (d_exp == 2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base;
    });
  } else if (d_exp == 3) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return base * base * base;
    });
  } else if (d_exp == -2) {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return 1.0 / (base * base);
    });
  } else {
    gpu_kernel(iter, [=]GPU_LAMBDA(Base_type base) -> Base_type {
      return pow_(base, exp);
    });
  }
}

void pow_tensor_scalar_kernel(TensorIteratorBase& iter, const Scalar& exp_scalar) {
  // Dispatch to fast specialization for sqrt, rsqrt and reciprocal
  if (!exp_scalar.isComplex()) {
    if (exp_scalar.equal(.5)) {
      return sqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-0.5)) {
      return rsqrt_kernel_cuda(iter);
    } else if (exp_scalar.equal(-1.0)) {
      return reciprocal_kernel_cuda(iter);
    }
  }
  if (isComplexType(iter.common_dtype()) || exp_scalar.isComplex()) {
    AT_DISPATCH_COMPLEX_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      gpu_kernel(iter, [=]GPU_LAMBDA(scalar_t base) -> scalar_t {
        return pow_(base, exp);
      });
    });
  } else if (isFloatingType(iter.common_dtype()) || exp_scalar.isIntegral(false)) {
    AT_DISPATCH_ALL_TYPES_AND2(kHalf, kBFloat16, iter.common_dtype(), "pow_cuda", [&]() {
      const auto exp = exp_scalar.to<scalar_t>();
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  } else {
    const auto exp = exp_scalar.to<float>();
    AT_DISPATCH_INTEGRAL_TYPES(iter.common_dtype(), "pow_cuda", [&]() {
      pow_tensor_scalar_kernel_impl<scalar_t>(iter, exp);
    });
  }
}

} // anonymous namespace

REGISTER_DISPATCH(pow_tensor_tensor_stub, &pow_tensor_tensor_kernel);
REGISTER_DISPATCH(pow_tensor_scalar_stub, &pow_tensor_scalar_kernel);

}} // namespace at::native
