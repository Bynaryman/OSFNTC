#include "hip/hip_runtime.h"
#include <ATen/AccumulateType.h>
#include <ATen/ATen.h>
#include <ATen/ceil_div.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/ThrustAllocator.h>
#include <ATen/native/sparse/cuda/SparseCUDAApplyUtils.cuh>
#include <ATen/native/cuda/SortingCommon.cuh>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseTensorUtils.h>
#include <c10/macros/Macros.h>
#include <c10/util/accumulate.h>

#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/gather.h>
#include <thrust/generate.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/transform.h>
#include <thrust/unique.h>
#include <thrust/system/cuda/execution_policy.h>
#include <thrust/binary_search.h>
#include <c10/macros/Macros.h>

namespace at { namespace native {

using namespace at::sparse;
using at::cuda::detail::TensorInfo;
using at::cuda::detail::getTensorInfo;

namespace {

template <typename scalar_t>
C10_LAUNCH_BOUNDS_1(1024)
__global__ void _sparse_mask_copy_kernel(
    int64_t total_threads,
    int64_t t_nnz,
    const TensorInfo<int64_t, int64_t> t_indices_ti,
    const TensorInfo<int64_t, int64_t> mask_indices_ti,
    const TensorInfo<int64_t, int64_t> t_indices_pos_ti,
    const TensorInfo<scalar_t, int64_t> t_values_ti,
    TensorInfo<scalar_t, int64_t> r_values_ti) {
  const int64_t i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= total_threads) return;
  const int64_t j = t_indices_pos_ti.data[i];

  bool has_match = false;
  if (j >= 0 &&  j < t_nnz && t_indices_ti.data[j] == mask_indices_ti.data[i]) {
    has_match = true;
  }

  int64_t values_stride0 = r_values_ti.strides[0];
  int64_t out_start = i * values_stride0;
  int64_t out_end = (i + 1) * values_stride0;
  int64_t in_start = j * t_values_ti.strides[0];

  if (has_match) {
    for (int64_t out_i = out_start, in_i = in_start; out_i < out_end; out_i++, in_i++) {
      r_values_ti.data[out_i] = t_values_ti.data[in_i];
    }
  }
}

} // end namespace

SparseTensor _coalesce_sparse_cuda(const SparseTensor& self) {
  int64_t nnz = self._nnz();
  TORCH_INTERNAL_ASSERT(!self.is_coalesced());
  // NOTE: Since `coalesce` is not an in-place operation when `is_coalesced` is false,
  // we should keep the original tensor intact and do coalesce on a copy of the tensor
  if (nnz < 2) {
    SparseTensor dst = self.clone();
    dst._coalesced_(true);
    return dst;
  }

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::cuda::ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);
  // Replace instances with

  // For indices, a simple sort + unique suffices
  // For values, we use a custom kernel for segmented reduction (can't use Thrust due to indirection).

  Tensor values = self._values();

  int64_t sparse_dim = self.sparse_dim();

  // indices will be modified by Thrust, so we have to clone or use new storage
  // here.
  Tensor indices1D = flatten_indices(self._indices(), self.sizes(), true);

  Tensor origIndices = at::empty({nnz}, self._indices().options());
  Tensor uniqueOffsets = at::empty({nnz}, self._indices().options());

  typedef thrust::device_ptr<int64_t> thrust_ptr;
  thrust_ptr indicesIter(indices1D.data_ptr<int64_t>());
  thrust_ptr origIndicesIter(origIndices.data_ptr<int64_t>());
  thrust_ptr uniqueOffsetsIter(uniqueOffsets.data_ptr<int64_t>());


  // Fill sortedOrigIndices with sequential indices
  thrust::counting_iterator<int64_t> countIterI(0);
  thrust::counting_iterator<int64_t> countIterO(0);

  thrust::copy(policy, countIterI, countIterI + nnz, origIndicesIter);
  thrust::copy(policy, countIterO, countIterO + nnz, uniqueOffsetsIter);

  thrust::sort_by_key(policy,
    indicesIter, indicesIter + nnz,
    origIndicesIter, LTOp<int64_t>()
  );

  // this forces device-host synchronization!
  thrust::pair<thrust_ptr, thrust_ptr> newEnd = thrust::unique_by_key(policy,
    indicesIter, indicesIter + nnz,
    uniqueOffsetsIter
  );
  int64_t newNnz = newEnd.first - indicesIter;

  indices1D.resize_({1, newNnz});
  auto newValues_size = values.sizes().vec();
  newValues_size[0] = newNnz;
  Tensor newValues = at::empty(newValues_size, values.options());

  // If there is no values to copy, save running the kernel.
  if (newValues.numel() > 0) {
    const int SZ = 4;
    values = values.contiguous();
    int64_t stride = c10::multiply_integers(values.sizes().slice(1));
    dim3 grid(ceil_div(newNnz, (int64_t) SZ), ceil_div(stride, (int64_t) C10_WARP_SIZE*SZ));
    dim3 block(C10_WARP_SIZE, SZ);
    AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND2(
      at::ScalarType::Half, at::ScalarType::BFloat16, values.scalar_type(), "coalesce_sparse_cuda", [&] {
        using cuda_accscalar_t = acc_type<scalar_t, /* is_cuda */ true>;
        apply::coalesceValuesKernel<scalar_t, cuda_accscalar_t><<<grid, block, 0, stream>>>(
          uniqueOffsets.data_ptr<int64_t>(),
          origIndices.data_ptr<int64_t>(),
          values.data_ptr<scalar_t>(),
          newValues.data_ptr<scalar_t>(),
          nnz,
          newNnz,
          stride
        );
        C10_CUDA_KERNEL_LAUNCH_CHECK();
      });
  }

// this grid-strided version is slower but probably more flexible
  // to different sizes
  // int64_t blockX = min(stride, (int64_t) 512);
  // dim3 block(blockX, 512 / blockX);
  // int64_t grid = min((int64_t) 1024, ceil_div((int64_t) newNnz * stride, (int64_t) block.x * block.y));
  // THCSTensor_coalesceValuesKernel_gridStrided<real, accreal><<<grid, block, 0, stream> >>(
  //   THCIndexTensor_(data)(state, uniqueOffsets),
  //   THCIndexTensor_(data)(state, origIndices),
  //   THCTensor_(data)(state, values),
  //   THCTensor_(data)(state, newValues),
  //   nnz,
  //   newNnz,
  //   stride
  // );
  // C10_CUDA_KERNEL_LAUNCH_CHECK();

  ////////////////////////////////////////////////////////////
  // unflatten indices if necessary
  Tensor newIndices;
  if (sparse_dim == 1) {
    newIndices = indices1D;
  } else {
    newIndices = at::empty({sparse_dim, newNnz}, origIndices.options());
    for (int64_t d = sparse_dim - 1; d >= 0; d--) {
      // NB: Not a select, so I can preserve the outer dimension
      Tensor indicesSlice = newIndices.narrow(0, d, 1);
      // Note for the porting guide: THCTensor_(copy) does NOT do normal
      // broadcasting logic; instead, it will blast the elements from one
      // to the other so long as the numel is the same
      indicesSlice.copy_(indices1D);
      indices1D.divide_(self.size(d), "trunc");
      indicesSlice.add_(indices1D, -self.size(d));
    }
  }
  ////////////////////////////////////////////////////////////
  // We can use unsafe sparse tensor constructor because the indices do not
  // need to be revalidated as we do not add or change indices, just remove
  // duplicates.
  SparseTensor dst = ::at::native::_sparse_coo_tensor_unsafe(newIndices, newValues, self.sizes())._coalesced_(true);

  AT_CUDA_CHECK(hipGetLastError());
  return dst;
}

Tensor sparse_mask_helper_cuda(
    const SparseTensor& t,
    const Tensor& mask_indices) {
  /*
    This is a helper function which filter values from `t._values()` using the
    `mask_indices`. This CUDA implementation uses `thrust::lower_bound`
    operation to find the intersection of the `mask_indices` and the
    `t._indices()` to then filter the values.

    Inputs:
      `t`             - coalesced sparse tensor input
      `mask_indices`  - mask indices tensor

    Note: The nnz in the output tensor will be same as the `mask_indices`. So it will
    works independently if the mask is coalesced or not.
  */
  TORCH_CHECK(t.is_sparse(), "t: input is not a sparse tensor");
  TORCH_CHECK(t.is_coalesced(), "t:  input is uncoalesced");
  TORCH_CHECK(mask_indices.dim() == t._indices().dim(), "mask_indices: operands have incompatible indices dim; self has dim ",
      t._indices().dim(), " but mask has dim ", mask_indices.dim());
  TORCH_CHECK(mask_indices.is_contiguous(), "mask_indices: mask is not contiguous");

  int64_t r_nnz = mask_indices.size(1);
  auto t_values = t._values().contiguous();
  auto full_size = t.sizes();
  auto vsize = t_values.sizes().vec();
  vsize[0] = r_nnz;


  if (t.sparse_dim() == 0) {
    Tensor t_values_expand = t_values;
    t_values_expand = t_values_expand.expand(vsize).contiguous();
    return t_values_expand;
  }
  Tensor r_values = at::zeros({vsize}, t_values.options());
  auto t_indices = t._indices().contiguous();
  auto t_nnz = t._nnz();

  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  at::cuda::ThrustAllocator allocator;
  auto policy = thrust::cuda::par(allocator).on(stream);

  // Step 1: flatten the sparse indices `t._indices()` tensor into a 1D indices
  // tensor `t_flatten_indices`.
  auto t_flatten_indices = at::sparse::flatten_indices(t_indices, full_size).contiguous();

  // Step 2: flatten the sparse indices `mask_indices` tensor into a 1D indices
  // tensor `mask_flatten_indices`. Note: This could be not sorted if the input
  // indices in the constructor are not in a coalesced form
  auto flattened_mask_indices =
      at::sparse::flatten_indices(mask_indices, full_size);

  Tensor t_indices_pos = at::empty({r_nnz}, mask_indices.options());

  // Step 3: Match the flattened `mask_indices` with the flattened
  // `t._indices()` using the `thrust::lower_bound`
  thrust::lower_bound(
      policy,
      t_flatten_indices.data_ptr<int64_t>(),
      t_flatten_indices.data_ptr<int64_t>() + t_nnz,
      flattened_mask_indices.data_ptr<int64_t>(),
      flattened_mask_indices.data_ptr<int64_t>() + r_nnz,
      t_indices_pos.data_ptr<int64_t>());

  // Step 4: Copy the Filtered `t._values()` using the matches at `t_indices_pos`
  if (r_nnz > 0 && t_values.numel() > 0) {
    int64_t block_size = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock, 1024);
    auto grid_size = ceil_div(r_nnz, block_size);

    auto t_indices_ti = getTensorInfo<int64_t, int64_t>(t_flatten_indices);
    auto mask_indices_ti =
        getTensorInfo<int64_t, int64_t>(flattened_mask_indices);
    auto t_indices_pos_ti =
        getTensorInfo<int64_t, int64_t>(t_indices_pos);

    AT_DISPATCH_FLOATING_AND_COMPLEX_TYPES_AND1(kHalf,
        r_values.scalar_type(), "sparse_mask_helper_cuda", [&] {
          auto t_values_ti = getTensorInfo<scalar_t, int64_t>(t_values);
          auto r_values_ti =
              getTensorInfo<scalar_t, int64_t>(r_values);

          _sparse_mask_copy_kernel<scalar_t><<<grid_size, block_size, 0, stream>>>(
              r_nnz,
              t_nnz,
              t_indices_ti,
              mask_indices_ti,
              t_indices_pos_ti,
              t_values_ti,
              r_values_ti);
          C10_CUDA_KERNEL_LAUNCH_CHECK();
        });
  }
  return r_values;
}
}} // namespace at::native
