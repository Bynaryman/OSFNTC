#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/ExpandUtils.h>
#include <ATen/InitialTensorOptions.h>
#include <ATen/NativeFunctions.h>
#include <ATen/SparseCsrTensorImpl.h>
#include <ATen/SparseCsrTensorUtils.h>
#include <ATen/SparseTensorUtils.h>
#include <ATen/WrapDimUtilsMulti.h>
#include <ATen/native/BinaryOps.h>
#include <ATen/native/Resize.h>
#include <algorithm>

#include <hip/hip_runtime.h>
#include <type_traits>


#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAUtils.h>
#include <ATen/cuda/ThrustAllocator.h>
#include <c10/cuda/CUDACachingAllocator.h>

#include <ATen/native/sparse/cuda/SparseBlasImpl.h>
#include <ATen/native/sparse/cuda/SparseCUDABlas.h>
#include <ATen/native/sparse/cuda/SparseCUDATensorMath.cuh>

#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/for_each.h>
#include <thrust/sequence.h>

namespace at {
namespace native {

namespace {

template <typename input_t, typename output_t>
__global__ void convert_indices_from_coo_to_csr_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t size, const int64_t numel) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;
  if (tid == 0) {
    for (int64_t i = 0; i <= data_in[0]; i++)
      data_out[i] = static_cast<output_t>(0);
  } else if (tid < numel) {
    for (int64_t i = data_in[tid - 1]; i < data_in[tid]; i++)
      data_out[i + 1] = static_cast<output_t>(tid);
  } else if (tid == numel) {
    for (int64_t i = data_in[numel - 1] + 1; i < size + 1; i++)
      data_out[i] = static_cast<output_t>(numel);
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_coo_to_csr_cuda(const Tensor& result, const Tensor& input, const int64_t size) {
  int64_t numel = input.numel();
  const input_t* data_in = input.data_ptr<input_t>();
  output_t* data_out = result.data_ptr<output_t>();

  if (numel == 0) {
    result.zero_();
    return;
  }

  // Run (numel + 1) threads...
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (numel + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  convert_indices_from_coo_to_csr_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, data_in, size, numel);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

template <typename input_t, typename output_t>
__global__ void convert_indices_from_csr_to_coo_cuda_kernel(output_t* data_out, const input_t* data_in, const int64_t nrows) {
  int64_t tid = blockDim.x * blockIdx.x + threadIdx.x;

  if (tid < nrows) {
    for (int64_t i = data_in[tid]; i < data_in[tid + 1]; i++)
      data_out[i] = static_cast<output_t>(tid);
  }
}

template <typename input_t, typename output_t>
void convert_indices_from_csr_to_coo_cuda(const Tensor& indices, const Tensor& crow_indices, const Tensor& col_indices) {
  int64_t nrows = crow_indices.numel() - 1;
  if (nrows == 0) {
    indices.zero_();
    return;
  }

  auto crow_indices_ = crow_indices.expect_contiguous();
  const input_t* crow_indices_data_in = crow_indices_->data_ptr<input_t>();
  TORCH_INTERNAL_ASSERT(indices.is_contiguous());
  output_t* data_out = indices.data_ptr<output_t>();

  // Run nrows threads...
  int64_t THREADS = at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock;
  int64_t BLOCKS = (nrows + THREADS) / THREADS;
  at::cuda::CUDAStream stream = at::cuda::getCurrentCUDAStream();
  indices.select(0, 1).copy_(*col_indices.expect_contiguous());
  convert_indices_from_csr_to_coo_cuda_kernel<<<BLOCKS, THREADS, 0, stream>>>(data_out, crow_indices_data_in, nrows);
  C10_CUDA_KERNEL_LAUNCH_CHECK();
}

} // namespace

using namespace at::sparse_csr;
// certain utiliy functions are usable from sparse COO.
using namespace at::sparse;

Tensor& add_out_dense_sparse_csr_cuda(
    Tensor& output,
    const Tensor& dense,
    const SparseCsrTensor& src,
    const Scalar& alpha) {
  TORCH_INTERNAL_ASSERT(dense.layout() == kStrided);
  TORCH_INTERNAL_ASSERT(src.is_sparse_csr());
  TORCH_INTERNAL_ASSERT(dense.is_cuda());

  TORCH_CHECK(
      output.is_contiguous(),
      "out argument must be contiguous, but got: ",
      output.suggest_memory_format());
  TORCH_CHECK(
      output.is_cuda(),
      "add: expected 'out' to be CUDA tensor, but got tensor on device: ",
      output.device());

  TORCH_CHECK(
      src.is_cuda(),
      "add: expected 'other' to be a CUDA tensor, but got tensor on device: ",
      src.device());

  TORCH_CHECK(
      dense.sizes().equals(src.sizes()),
      "add: expected 'self' and 'other' to have same size, but self has size ",
      dense.sizes(),
      " while other has size ",
      src.sizes(),
      " (FYI: dense-sparse addition does not currently support broadcasting)");

  auto commonDtype = promoteTypes(dense.scalar_type(), src.scalar_type());
  TORCH_CHECK(
      canCast(commonDtype, output.scalar_type()),
      "Can't convert result type ",
      commonDtype,
      " to output ",
      output.scalar_type(),
      " in add operation");

  Tensor src_values = src.values();
  Tensor src_crow_indices = src.crow_indices();
  Tensor src_col_indices = src.col_indices();

  resize_output(output, dense.sizes());

  Tensor resultBuffer = output;
  Tensor valuesBuffer = src_values.to(commonDtype);
  if (output.scalar_type() != commonDtype) {
    resultBuffer = dense.to(commonDtype);
  } else if (!is_same_tensor(output, dense)) {
    resultBuffer.copy_(dense);
  }
  AT_DISPATCH_ALL_TYPES_AND_COMPLEX_AND3(
      kHalf, kBool, kBFloat16,
      commonDtype,
      "add_out_op2_sparse_csr",
      [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
        AT_DISPATCH_INDEX_TYPES(
            src_crow_indices.scalar_type(),
            "csr_add_out_crow_indices",
              [&valuesBuffer, &resultBuffer, &alpha, &src_crow_indices, &src_col_indices]() {
                scalar_t* values_accessor = valuesBuffer.data_ptr<scalar_t>();
                scalar_t* out_ptr = resultBuffer.data_ptr<scalar_t>();
                scalar_t cast_value = alpha.to<scalar_t>();

                index_t* crow_indices_accessor = src_crow_indices.data_ptr<index_t>();
                index_t* col_indices_accessor = src_col_indices.data_ptr<index_t>();
                int64_t out_storage_offset = resultBuffer.storage_offset();

                auto out_strides = resultBuffer.strides();
                int64_t out_strides0 = out_strides[0];
                int64_t out_strides1 = out_strides[1];

                hipStream_t stream = at::cuda::getCurrentCUDAStream();
                at::cuda::ThrustAllocator allocator;
                auto policy = thrust::cuda::par(allocator).on(stream);

               // Note that this could be wildly imbalanced if the sparsity pattern varies a lot between rows.
               thrust::for_each(
                    policy,
                    thrust::make_counting_iterator(int64_t(0)),
                    thrust::make_counting_iterator(int64_t(src_crow_indices.size(0) - 1)),
                    [values_accessor,
                    crow_indices_accessor,
                    col_indices_accessor,
                    out_ptr,
                    out_storage_offset,
                    out_strides0,
                    cast_value,
                    out_strides1
                    ]__device__(int64_t irow) {
                        index_t start_index = crow_indices_accessor[irow];
                        index_t end_index = crow_indices_accessor[irow + 1];

                        for (index_t i = start_index; i < end_index; ++i) {
                            auto icol = col_indices_accessor[i];
                            auto index = out_storage_offset + irow * out_strides0 + icol * out_strides1;
                            out_ptr[index] += cast_value * values_accessor[i];
                        }
                    });
              });
      });
  if (output.scalar_type() != commonDtype) {
    output.copy_(resultBuffer);
  }
  return output;
}

Tensor& add_out_sparse_csr_cuda(
    const Tensor& self,
    const SparseCsrTensor& other,
    const Scalar& alpha,
    SparseCsrTensor& out) {
  if (self.layout() == kStrided) {
    add_out_dense_sparse_csr_cuda(out, self, other, alpha);
  } else {
    TORCH_CHECK(
        self.sizes().equals(other.sizes()),
        "torch.add: Expected input tensors to have the same shape, but got tensor `self` with shape ",
        self.sizes(),
        " and tensor `other` with shape ",
        other.sizes());
    at::native::resize_as_sparse_csr_(out, self);
    sparse::impl::cuda::add_out_sparse_csr(self, other, Scalar(1), alpha, out);
  }
  return out;
}

TORCH_IMPL_FUNC(_convert_indices_from_coo_to_csr_structured_cuda) (
  const Tensor& input, const int64_t size, const bool out_int32, const Tensor& result
) {
  if (out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int>(result, input, size);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(input.scalar_type(), "convert_indices_from_coo_to_csr_cuda", [&] {
      convert_indices_from_coo_to_csr_cuda<scalar_t, int64_t>(result, input, size);
    });
  }
}

TORCH_IMPL_FUNC(_convert_indices_from_csr_to_coo_structured_cuda) (
  const Tensor& crow_indices, const Tensor& col_indices, const bool out_int32, const Tensor& result
) {
  if (out_int32) {
    AT_DISPATCH_INTEGRAL_TYPES(crow_indices.scalar_type(), "convert_indices_from_csr_to_coo_cuda", [&] {
      convert_indices_from_csr_to_coo_cuda<scalar_t, int32_t>(result, crow_indices, col_indices);
    });
  } else {
    AT_DISPATCH_INTEGRAL_TYPES(crow_indices.scalar_type(), "convert_indices_from_csr_to_coo_cuda", [&] {
      convert_indices_from_csr_to_coo_cuda<scalar_t, int64_t>(result, crow_indices, col_indices);
    });
  }
}

} // namespace native
} // namespace at
