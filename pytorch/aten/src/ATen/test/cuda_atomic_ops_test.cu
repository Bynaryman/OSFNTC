#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <ATen/cuda/Atomic.cuh>
#include <c10/test/util/Macros.h>
#include <c10/cuda/CUDAException.h>

#include <cmath>

constexpr int blocksize = 256;
constexpr int factor = 4;
constexpr int arraysize = blocksize / factor;

template <typename T>
__global__ void addition_test_kernel(T * a, T * sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = (tid) % arraysize;

  gpuAtomicAdd(&sum[idx], a[idx]);
}

template <typename T>
__global__ void mul_test_kernel(T * a, T * sum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int idx = (tid) % arraysize;

  gpuAtomicMul(&sum[idx], a[idx]);
}

template <typename T>
void test_atomic_add() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  for (int i = 0; i < arraysize; ++i) {
    a[i] = 1;
    sum[i] = 0;
    answer[i] = factor;
  }

  hipMalloc((void**)&ad, arraysize * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  addition_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

template <typename T>
void test_atomic_mul() {
  dim3 dimBlock(blocksize, 1);
  dim3 dimGrid(1, 1);

  T *ad, *sumd;

  std::vector<T> a(arraysize);
  std::vector<T> sum(arraysize);
  std::vector<T> answer(arraysize);

  for (int i = 0; i < arraysize; ++i) {
    a[i] = 2;
    sum[i] = 2;
    answer[i] = pow(sum[i], static_cast<T>(factor));
  }

  hipMalloc((void**)&ad, arraysize * sizeof(T));
  hipMalloc((void**)&sumd, arraysize * sizeof(T));

  hipMemcpy(ad, a.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(sumd, sum.data(), arraysize * sizeof(T), hipMemcpyHostToDevice);

  mul_test_kernel<<<dimGrid, dimBlock>>>(ad, sumd);
  C10_CUDA_KERNEL_LAUNCH_CHECK();

  hipMemcpy(sum.data(), sumd, arraysize * sizeof(T), hipMemcpyDeviceToHost);

  for (int i = 0; i < arraysize; ++i) {
    ASSERT_EQ(sum[i], answer[i]) << typeid(T).name();
  }

  hipFree(ad);
  hipFree(sumd);
}

TEST(TestAtomicOps, TestAtomicAdd) {
  test_atomic_add<uint8_t>();
  test_atomic_add<int8_t>();
  test_atomic_add<int16_t>();
  test_atomic_add<int32_t>();
  test_atomic_add<int64_t>();

  test_atomic_add<at::BFloat16>();
  test_atomic_add<at::Half>();
  test_atomic_add<float>();
  test_atomic_add<double>();
  test_atomic_add<c10::complex<float> >();
  test_atomic_add<c10::complex<double> >();
}

TEST(TestAtomicOps, DISABLED_ON_WINDOWS(TestAtomicMul)) {
  test_atomic_mul<at::BFloat16>();
  test_atomic_mul<at::Half>();
  test_atomic_mul<float>();
  test_atomic_mul<double>();
}
